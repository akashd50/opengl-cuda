#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include "headers/CudaKernelUtils.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <vector_functions.h>

//----------OPERATORS---------------------------------------------------------------------------------------------------

__device__ __host__ uchar4 operator+(const uchar4 &a, const uchar4 &b) {
    return make_uchar4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.z);
}

__device__ __host__ uchar4 operator*(const uchar4 &a, const float &b) {
    return make_uchar4(min(int(a.x*b), 255), min(int(a.y*b), 255), min(int(a.z*b), 255), a.w);
}

__device__ __host__ uchar4 operator/(const uchar4 &a, const float b) {
    return make_uchar4(int(a.x/b), int(a.y/b), int(a.z/b), 255);
}

__device__ __host__ float3 operator*(const float3 &a, const float &b) {
    return make_float3(a.x*b, a.y*b, a.z*b);
}

__device__ __host__ float3 operator*(const float &a, const float3 &b) {
    return b * a;
}

__device__ __host__ float3 operator/(const float3 &a, const float &b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

__device__ __host__ float3 operator/(const float a, const float3 &b) {
    return make_float3(a/b.x, a/b.y, a/b.z);
}

__device__ __host__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ __host__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ __host__ float3 operator-(const float a, const float3 &b) {
    return make_float3(a-b.x, a-b.y, a-b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float3 &b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

__device__ __host__ void add(float3 &a, float3 &b) {
    a.x += b.x; a.y += b.y; a.z += b.z;
}

__device__ __host__ void subtract(float3 &a, float3 &b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z;
}

__device__ __host__ void multiply(float3 &a, float b) {
    a.x *= b; a.y *= b; a.z *= b;
}

//----------VECTOR--OPERATIONS------------------------------------------------------------------------------------------

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ uchar4 toRGBA(const float3 &a) {
    return make_uchar4(int(a.x * 255), int(a.y * 255), int(a.z * 255), 255);
}

__device__ __host__ float3 t_to_vec(Ray &ray, float &t) {
    return ray.origin + (t * ray.direction);
}

__device__ float magnitude(float3 &a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ float len_squared(float3 a) {
    return a.x * a.x + a.y * a.y + a.z * a.z;
}

__device__ float3 normalize(float3 a) {
    float mag = magnitude(a);
    return make_float3(a.x, a.y, a.z)/mag;
}

__device__ void normalizeR(float3 &a) {
    float mag = magnitude(a);
    a.x = a.x/mag; a.y = a.y/mag; a.z = a.z/mag;
}

__device__ float3 cross(float3 a, float3 b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

__device__ float3 clamp(float3 a, float min, float max) {
    float x = a.x; float y = a.y; float z = a.z;
    x = x < min ? min : x; x = x > max ? max : x;
    y = y < min ? min : y; y = y > max ? max : y;
    z = z < min ? min : z; z = z > max ? max : z;
    return make_float3(x, y, z);
}

//----------RT-FUNCTIONS------------------------------------------------------------------------------------------------

__device__ float3 cast_ray(int &x, int &y, int &width, int &height) {
    float d = 1.0;
    float fov = 60.0;
    float aspect_ratio = ((float)width) / ((float)height);
    float h = d * (float)tan((3.1415 * fov) / 180.0 / 2.0);
    float w = h * aspect_ratio;

    float top = h;
    float bottom = -h;
    float left = -w;
    float right = w;

    float u = left + (right - left) * float(x) / ((float)width);
    float v = bottom + (top - bottom) * (((float)height) - float(y)) / ((float)height);
    return make_float3(u, v, -d);
}

__device__ float3 getReflectedRay(float3 &direction, float3 &normal) {
    float3 ray_dir = normalize(direction);
    return ray_dir - 2.0f * normal * dot(ray_dir, normal);
}

__device__ float3 getSphereNormal(float3 &point, CudaSphere* sphere) {
    float3 normal = point - sphere->position;
    return normalize(normal);
}

__device__ float3 getTriangleNormal(CudaTriangle &triangle) {
    return normalize(cross(triangle.b - triangle.a, triangle.c - triangle.a));
}

__device__ float checkHitOnPlane(float3 &e, float3 &d, float3 &center, float3 &normal) {
    /*Checks the hit on an infinite plane for the given normal and returns t value*/
    float denominator = dot(normal, d);
    if (denominator != 0.0) {
        float t = dot(normal, (center - e)) / denominator;
        return t;
    }
    return MAX_T;
}

__device__ bool checkHitOnTriangle(Ray &ray, CudaTriangle &triangle, HitInfo &hitInfo) {
    /*Checks the hit on the triangle and returns t value. I first use the plane hit and then check if its inside the triangle*/
    float3 normal = getTriangleNormal(triangle);
    float t = checkHitOnPlane(ray.origin, ray.direction, triangle.a, normal);

    if (t == MAX_T) return false;

    float3 x = t_to_vec(ray, t);
    float aTest = dot(cross(triangle.b - triangle.a, x - triangle.a), normal);
    float bTest = dot(cross(triangle.c - triangle.b, x - triangle.b), normal);
    float cTest = dot(cross(triangle.a - triangle.c, x - triangle.c), normal);
    bool isValidHit = ((aTest >= 0 - HIT_T_OFFSET_1 && bTest >= 0 - HIT_T_OFFSET_1 && cTest >= 0 - HIT_T_OFFSET_1)
                   || (aTest <= 0 + HIT_T_OFFSET_1 && bTest <= 0 + HIT_T_OFFSET_1 && cTest <= 0 + HIT_T_OFFSET_1));
    if (isValidHit && t >= HIT_T_OFFSET && t < hitInfo.t) {
        hitInfo.t = t;
        hitInfo.point = t_to_vec(ray, hitInfo.t);
        hitInfo.normal = normal;
        return true;
    }
    return false;
}

__device__ bool check_hit_on_sphere(Ray &ray, CudaSphere* sphere, HitInfo &hitInfo, bool debug) {
    float3 center_2_eye = ray.origin - sphere->position;
    float ray_dot_ray = dot(ray.direction, ray.direction);
    float ray_dot_c2e = dot(ray.direction, center_2_eye);
    float discriminant = pow(ray_dot_c2e, 2) - ray_dot_ray * (dot(center_2_eye, center_2_eye) - pow(sphere->radius, 2));
    float mint = MAX_T;
    if (discriminant > 0) {
        discriminant = sqrt(discriminant);
        float init = -ray_dot_c2e;
        float t1 = (init + discriminant) / ray_dot_ray;
        float t2 = (init - discriminant) / ray_dot_ray;

        mint = min(t1, t2);
        if (mint < HIT_T_OFFSET) {
            mint = max(t1, t2);
        }
    }
    else if (discriminant == 0) {
        float init = -ray_dot_c2e;
        mint = init / ray_dot_ray;
    }

    if (mint >= HIT_T_OFFSET && mint < hitInfo.t) {
        hitInfo.t = mint;
        hitInfo.object = sphere;
        hitInfo.point = t_to_vec(ray, hitInfo.t);
        hitInfo.normal = getSphereNormal(hitInfo.point, sphere);
        hitInfo.objectId = sphere->id;
        return true;
    }
    return false;
}

__device__ void printBounds(Bounds* bounds) {
    printf("AABB (%0.2f, %0.2f, %0.2f, %0.2f, %0.2f, %0.2f)", bounds->top, bounds->bottom,
           bounds->left, bounds->right, bounds->front, bounds->back);
}

__device__ void print2DUtil(BVHBinaryNode *root, int space)
{
    // Base case
    if (root == nullptr)
        return;

    // Increase distance between levels
    space += 10;

    // Process right child first
    print2DUtil(root->right, space);

    // Print current node after space
    // count
    printf("\n");
    for (int i = 10; i < space; i++)
        printf(" ");
    //print data
    printf("[{");
    //if (root->numObjects == 0) {
    //printBounds(root->bounds);
    //}
    printf("} ");
    for (int i=0; i<root->numObjects; i++) {
        printf("%d, ", root->objectsIndex[i]);
    }
    printf("]");
    //cout<<root->data<<"\n";

    // Process left child
    print2DUtil(root->left, space);
}

__device__ void swap(float &a, float &b) {
    float t = a;
    a = b;
    b = t;
}

/*
 * float3 invDir = 1.0 / d;
    //float3 invDir = make_float3(0.0, 0.0, 0.0) - d;

    float xVal = (invDir.x < 0) ? bounds->right : bounds->left;
    float yVal = (invDir.y < 0) ? bounds->top : bounds->bottom;
    float zVal = (invDir.z < 0) ? bounds->front : bounds->back;
    float tmin, tmax, tymin, tymax, tzmin, tzmax;

    tmin = (xVal - e.x) * invDir.x;
    tmax = (xVal - e.x) * invDir.x;
    tymin = (yVal - e.y) * invDir.y;
    tymax = (yVal - e.y) * invDir.y;

    if ((tmin > tymax) || (tymin > tmax))
        return MAX_T;
    if (tymin > tmin)
        tmin = tymin;
    if (tymax < tmax)
        tmax = tymax;

    tzmin = (zVal - e.z) * invDir.z;
    tzmax = (zVal - e.z) * invDir.z;

    if ((tmin > tzmax) || (tzmin > tmax))
        return MAX_T;
    if (tzmin > tmin)
        tmin = tzmin;
    if (tzmax < tmax)
        tmax = tzmax;
 */

__device__ MinMaxT checkHitOnAABB(Ray &ray, Bounds* bounds, bool debug) {
    if (bounds == nullptr) return {MIN_T, MAX_T};

    float tmin = (bounds->left - ray.origin.x) / ray.direction.x;
    float tmax = (bounds->right - ray.origin.x) / ray.direction.x;

    if (tmin > tmax) swap(tmin, tmax);

    float tymin = (bounds->bottom - ray.origin.y) / ray.direction.y;
    float tymax = (bounds->top - ray.origin.y) / ray.direction.y;

    if (tymin > tymax) swap(tymin, tymax);

    if ((tmin > tymax) || (tymin > tmax))
        return {MIN_T, MAX_T};

    if (tymin > tmin)
        tmin = tymin;

    if (tymax < tmax)
        tmax = tymax;

    float tzmin = (bounds->back - ray.origin.z) / ray.direction.z;
    float tzmax = (bounds->front - ray.origin.z) / ray.direction.z;

    if (tzmin > tzmax) swap(tzmin, tzmax);

    if ((tmin > tzmax) || (tzmin > tmax))
        return {MIN_T, MAX_T};

    if (tzmin > tmin)
        tmin = tzmin;

    if (tzmax < tmax)
        tmax = tzmax;

    return {tmin != tmin ? 0 : tmin, tmax != tmax ? 0 : tmax};
}

__device__ bool checkHitOnNodeTriangles(Ray &ray, BVHBinaryNode* node, CudaMesh* mesh, HitInfo &hitInfo, bool debug) {
    bool tUpdated = false;
    if (node != nullptr && node->numObjects != 0) { // Is a leaf node
        for (int j=0; j<node->numObjects; j++) {
            int objIndex = node->objectsIndex[j];
            CudaTriangle triangle = mesh->triangles[objIndex];
            if (checkHitOnTriangle(ray, triangle, hitInfo)) {
                tUpdated = true;
                if (debug) {
                    printf("New hit on triangle at (%d) MinT (%f)\n", node->objectsIndex[j], hitInfo.t);
                }
            }
        }
    }

    return tUpdated;
}

__device__ unsigned long numMeshHitChecks = 0;
__device__ unsigned long numOverallTriangleChecks = 0;

__device__ bool checkHitOnMeshHelperNR(Ray &ray, CudaMesh* mesh, HitInfo &hitInfo, bool debug) {
    if (debug) {
        printf("\n\nStarting to check hit on Mesh\n");
    }

    int numTrianglesChecked = 0;
    int numAABBChecks = 0;
    numMeshHitChecks++;

    auto stack = new Stack<BVHBinaryNode*>();
    if (!stack->init()) return false;

    BVHBinaryNode* curr = mesh->bvhRoot;

    bool tUpdated = false;
    while (!stack->empty() || curr != nullptr) {
        if (curr != nullptr) {
            auto currT = checkHitOnAABB(ray, curr->bounds, debug);
            numAABBChecks++;
            /*if (debug) {
                printf("AABB Checking curr MinT(%f) MaxT(%f) HitT(%f)\n", currT.minT, currT.maxT, hitInfo.t);
                printBounds(curr->bounds);
                printf("\n");
            }*/

            if (currT.minT != MIN_T && currT.maxT != MAX_T
            && currT.minT <= hitInfo.t && currT.maxT >= 0) {
                numTrianglesChecked += curr->numObjects;
                numOverallTriangleChecks += curr->numObjects;
                if (checkHitOnNodeTriangles(ray, curr, mesh, hitInfo, debug)) {
                    tUpdated = true;
                    hitInfo.object = mesh;
                    hitInfo.objectId = mesh->id;
                }

                stack->push(curr);
                curr = curr->left;
            } else {
                curr = nullptr;
            }
        } else {
            curr = stack->top();
            stack->pop();
            curr = curr->right;
        }
    }
    if (debug) printf("\nNum Hit Checked: Triangles(%d) | AABB(%d)\n", numTrianglesChecked, numAABBChecks);

    stack->clean();
    free(stack);
    return tUpdated;
}

__device__ int wang_hash(int seed) {
    seed = (seed ^ 61) ^ (seed >> 16);
    seed *= 9;
    seed = seed ^ (seed >> 4);
    seed *= 0x27d4eb2d;
    seed = seed ^ (seed >> 15);
    return seed;
}

__device__ void randFloat3(hiprandState &randState, float3 &vec) {
    vec.x = hiprand_normal(&randState);
    vec.y = hiprand_normal(&randState);
    vec.z = hiprand_normal(&randState);
}

__device__ void randFloat3InMesh(hiprandState &randState, CudaMesh* mesh, float3 &vec) {
    randFloat3(randState, vec);
    normalizeR(vec);
    vec.x = mesh->position.x + vec.x * mesh->dimensions.x;
    vec.y = mesh->position.y + vec.y * mesh->dimensions.y;
    vec.z = mesh->position.z + vec.z * mesh->dimensions.z;
//    vec.x = mesh->position.x + vec.x;
//    vec.y = mesh->position.y + vec.y;
//    vec.z = mesh->position.z + vec.z;
}

__device__ float3 getReflectedDiffuseRay(HitInfo &hitInfo, CudaThreadData &threadData, bool useReflected) {
    float3 p_n = hitInfo.point + (useReflected ? hitInfo.reflected * 1.5 : hitInfo.normal * 1.5);
    float3 vec = make_float3(0, 0, 0);
    randFloat3(threadData.randState, vec);

    normalizeR(vec);
    if (useReflected) {
        vec = vec * hitInfo.object->material->roughness;
    }

    float3 p = p_n + vec;
    return normalize(p - hitInfo.point);
}

__device__ float3 ray_color(const Ray& ray) {
    float t = 0.5f * (ray.direction.y + 1.0f);
    return (1.0f - t) * make_float3(0.5, 0.7, 1.0) + t * make_float3(1.0, 1.0, 1.0);
}

__device__ void doHitTest(Ray &ray, HitInfo &hitInfo, CudaThreadData &threadData) {
    CudaScene* scene = threadData.scene;
    for (int i=0; i<scene->numObjects; i++) {
        if (scene->objects[i]->type == SPHERE) {
            auto sphere = (CudaSphere*)scene->objects[i];
            bool isHit = check_hit_on_sphere(ray, sphere, hitInfo, threadData.debug);
            if (isHit && threadData.debug) {
                printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
            }
        }
        else if (scene->objects[i]->type == MESH) {
            auto mesh = (CudaMesh*)scene->objects[i];
            bool isHit = checkHitOnMeshHelperNR(ray, mesh, hitInfo, threadData.debug);
            if (isHit && threadData.debug) {
                printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
            }
        }
    }

    for (int i=0; i<scene->numLights; i++) {
        auto light = (CudaLight*)scene->lights[i];
        if (light->lightType == SKYBOX_LIGHT) {
            auto sphere = ((CudaSkyboxLight*)scene->lights[i])->sphere;
            if (check_hit_on_sphere(ray, sphere, hitInfo, threadData.debug)) {
                hitInfo.object = light;
                hitInfo.lighting = ray_color(ray);
                hitInfo.objectId = light->id;
                if (threadData.debug) {
                    printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
                }
            }
        } else if (light->lightType == MESH_LIGHT) {
            auto mesh = ((CudaMeshLight*)scene->lights[i])->mesh;
            if (checkHitOnMeshHelperNR(ray, mesh, hitInfo, threadData.debug)) {
                hitInfo.object = light;
                hitInfo.lighting = ((CudaMeshLight*)scene->lights[i])->color;
                hitInfo.objectId = light->id;
                if (threadData.debug) {
                    printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
                }
            }
        }
    }
}

__device__ void getLighting(HitInfo &hitInfo, CudaThreadData &threadData) {
    //    To Be Used Later when more lights are added

    CudaScene* scene = threadData.scene;
    HitInfo tempHitInfo;
    int numSamplesPerLight = 1;
    for (int i=0; i<scene->numLights; i++) {
        auto light = (CudaLight *) scene->lights[i];
        if (light->lightType == SKYBOX_LIGHT) {
            for (int s=0; s<numSamplesPerLight; s++) {
                float3 randVecToLight = getReflectedDiffuseRay(hitInfo, threadData, false);
                tempHitInfo.t = MAX_T;

                Ray newRay(hitInfo.point, randVecToLight);
                doHitTest(newRay, tempHitInfo, threadData);
                if (tempHitInfo.objectId == light->id) {
                    hitInfo.lighting = hitInfo.lighting + ray_color(newRay) * light->intensity;
                }
            }
        }
        // Maybe only do lighting here and not material stuff - so add them separately in the main trace ray func..
        if (light->lightType == MESH_LIGHT) {
            CudaMeshLight* meshLight = (CudaMeshLight*)light;
            float3 randVecToLight = make_float3(0, 0, 0);
            for (int s=0; s<numSamplesPerLight; s++) {
                randFloat3InMesh(threadData.randState, meshLight->mesh, randVecToLight);
                randVecToLight = normalize(randVecToLight - hitInfo.point);
                tempHitInfo.t = MAX_T;

                Ray newRay(hitInfo.point, randVecToLight);
                doHitTest(newRay, tempHitInfo, threadData);
                if (tempHitInfo.objectId == light->id) {
                    float diff = max(dot(hitInfo.normal, randVecToLight), 0.0f);
                    hitInfo.lighting = hitInfo.lighting + diff * meshLight->color * meshLight->intensity;

//                    float3 viewDir = normalize(viewPos - FragPos);
//                    vec3 reflectDir = reflect(-lightDir, norm);
                }
            }
        }
    }
    hitInfo.lighting = hitInfo.lighting/(float)(numSamplesPerLight * scene->numLights);
}

__device__ void doPathTracing(Ray &ray, HitInfo &hitInfo, CudaThreadData &threadData, float3 &lighting, int &index) {
    if (index >= 3) return;

    if (index != 0) {
        hitInfo.reflected = normalize(getReflectedRay(ray.direction, hitInfo.normal));
        ray.direction = getReflectedDiffuseRay(hitInfo, threadData, true);
    } else {
        ray.direction = getReflectedDiffuseRay(hitInfo, threadData, false);
    }

    hitInfo.t = MAX_T;

    ray.origin = hitInfo.point;
    doHitTest(ray, hitInfo, threadData);
    if (hitInfo.object->type == LIGHT) {
        lighting = lighting + hitInfo.lighting;
    } else {
        index++;
        CudaMaterial* mat = hitInfo.object->material;
        doPathTracing(ray, hitInfo, threadData, lighting, index);
        //Calculate the lighting from all other lights here
        //getLighting(ray, hitInfo, threadData, lighting);
        lighting = lighting * mat->albedo * mat->diffuse;
    }
}

__device__ void calculateLightingPathTraced(HitInfo &hitInfo, CudaThreadData &threadData) {
    if (hitInfo.object->type == LIGHT) {
        // May change this to include other light properties like intensity and stuff
        return;
    }

    float3 lighting = make_float3(0, 0, 0);
    Ray emptyRay(make_float3(0, 0, 0), make_float3(0, 0, 0));
    HitInfo newHit(hitInfo);
    int index = 0;
    doPathTracing(emptyRay, newHit, threadData, lighting, index);
    hitInfo.lighting = lighting * hitInfo.object->material->diffuse;
}

__device__ float3 traceSingleRay(Ray &ray, int maxBounces, CudaThreadData &threadData) {
    auto stack = new Stack<HitInfo>();
    stack->init();

//    float3 newRay = ray.direction;
//    float3 newEye = eye;
    Ray currRay = ray;
    int bounceIndex = 0;
    while(bounceIndex < maxBounces) {
        HitInfo hitInfo;
        doHitTest(currRay, hitInfo, threadData);
        if (hitInfo.isHit()) {
            stack->push(hitInfo);
            if (hitInfo.object->type >= LIGHT) {
                break;
            }

            hitInfo.reflected = normalize(getReflectedRay(ray.direction, hitInfo.normal));
            currRay = Ray(hitInfo.point, getReflectedDiffuseRay(hitInfo, threadData, true));
            if (threadData.debug) {
                printf("\n\nHitInfo(%d); Hit T(%f) @ (%f, %f, %f) | Normal(%f, %f, %f) | Reflected(%f, %f, %f)\n\n\n\n\n\n\n",
                       hitInfo.objectId, hitInfo.t, hitInfo.point.x, hitInfo.point.y, hitInfo.point.z,
                       hitInfo.normal.x, hitInfo.normal.y, hitInfo.normal.z,
                       currRay.direction.x, currRay.direction.y, currRay.direction.z);
            }
        } else {
            break;
        }
        bounceIndex++;
    }

    if (threadData.debug) printf("\n\n\n\n\n\n\n\nStarting to Sum lighting\n\n");
    // Sum all colors from stack
    float3 color = make_float3(0.0, 0.0, 0.0);
    float3 currLighting = make_float3(0, 0, 0);
    if (stack->size() >= 2) {
        while(!stack->empty()) {
            HitInfo curr = stack->top();
            stack->pop();
//            if (debug) {
//                printf("\n\nCurr | Index(%d)\n", curr.index);
//                printf("Next | Index(%d)\n", curr.index);
//                printf("Color(%f, %f, %f)\n", lighting.x, lighting.y, lighting.z);
//            }
            if (curr.object->type < LIGHT) {
                //float3 currLighting = calculateLightingPathTraced(curr, threadData);
                getLighting(curr, threadData);
                currLighting = (currLighting + curr.lighting) * curr.object->material->diffuse;
                color = currLighting + (curr.object->material->reflective * color);
                currLighting = currLighting * curr.object->material->albedo;
            } else {
                //currLighting = curr.lighting;
                color = curr.lighting;
            }
        }
    } else if (stack->size() == 1) {
        HitInfo curr = stack->top(); stack->pop();
        //lighting = lighting + calculateLightingPathTraced(curr, threadData);
        color = curr.lighting;
    }

    color.x = sqrt(color.x);
    color.y = sqrt(color.y);
    color.z = sqrt(color.z);
    color = clamp(color, 0.0, 0.99);

    stack->clean();
    free(stack);

    return color;
}

__global__ void kernel_traceRays(hipSurfaceObject_t image, CudaScene* scene,  int startRowIndex, int startColIndex, int sampleIndex)
{
    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    int x = startColIndex + (int)threadIdx.x;
    int y = startRowIndex + (int)blockIdx.x;
    int randIndex = wang_hash(x * y) % scene->width;

    hiprandState state;
    hiprand_init (x*y*(sampleIndex+2), 0, 0, &state);

    CudaThreadData threadData;
    threadData.debug = false;
    threadData.randState = state;
    threadData.randIndex = randIndex;
    threadData.scene = scene;

    int maxBounces = 8;
    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, scene->width, scene->height) - eye;
    Ray initialRay(eye, ray);

    float3 sampledColor = make_float3(0, 0, 0);
    float p = 0.0005f /* hiprand_normal(&threadData.randState)*/;
    //int numSamples = 8; //i.e. 8 * 4 = 32 or 1 * 4 = 4
    //for(int i=0; i<numSamples; i++) {
    initialRay.direction = initialRay.direction + make_float3(p, p, 0);
    sampledColor = sampledColor + traceSingleRay(initialRay, maxBounces, threadData);
    initialRay.direction = initialRay.direction + make_float3(-p, -p, 0);
    sampledColor = sampledColor + traceSingleRay(initialRay, maxBounces, threadData);
    initialRay.direction = initialRay.direction + make_float3(-p, p, 0);
    sampledColor = sampledColor + traceSingleRay(initialRay, maxBounces, threadData);
    initialRay.direction = initialRay.direction + make_float3(p, -p, 0);
    sampledColor = sampledColor + traceSingleRay(initialRay, maxBounces, threadData);
    //}
    //uchar4 lighting = toRGBA(sampledColor/((float)numSamples * 4.0f));
    uchar4 color = toRGBA(sampledColor/4.0f);

//    float3 cVal = traceSingleRay(eye, ray, maxBounces, threadData);
//    uchar4 lighting = toRGBA(cVal);
    if (sampleIndex != 0) {
        uchar4 ec;
        surf2Dread(&ec, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
        color = make_uchar4((color.x + ec.x)/2.0f, (color.y + ec.y)/2.0f, (color.z + ec.z)/2.0f, 255);
    }
    surf2Dwrite(color, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
}

__global__ void kernel_traceSingleRay(hipSurfaceObject_t image, int x, int y, CudaScene* scene)
{
    int randIndex = (x * y) % scene->width;
    int maxBounces = 2;

    CudaThreadData threadData;
    hiprandState state;
    hiprand_init (x*y, 0, 0, &state);
    threadData.debug = true;
    threadData.randState = state;
    threadData.randIndex = randIndex;
    threadData.scene = scene;

    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, scene->width, scene->height) - eye;
    Ray initialRay(eye, ray);
    printf("\n\nRay (%f, %f, %f)\n", ray.x, ray.y, ray.z);
    uchar4 color = toRGBA(traceSingleRay(initialRay, maxBounces, threadData));
    printf("Final Color: (%d, %d, %d, %d)\n", color.x, color.y, color.z, color.w);
    printf("Overall NumTriangleHits(%lu) NumMeshHitChecks(%lu)\n", numOverallTriangleChecks, numMeshHitChecks);
    printf("Overall average triangle hits: (%f)\n", (float)numOverallTriangleChecks/(float)numMeshHitChecks);

    uchar4 ec;
    surf2Dread(&ec, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
    printf("\nExisting lighting (%d, %d, %d, %d)\n", ec.x, ec.y, ec.z, ec.w);
    printf("\nNew lighting (%d, %d, %d, %d)\n", color.x, color.y, color.z, color.w);
    color = make_uchar4((color.x + ec.x)/2.0f, (color.y + ec.y)/2.0f, (color.z + ec.z)/2.0f, 255);
    printf("\nCombined lighting (%d, %d, %d, %d)\n", color.x, color.y, color.z, color.w);

    surf2Dwrite(color, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
}

__device__ uchar4 getColorAt(hipSurfaceObject_t image, int2 imageDim, int2 index) {
    if (index.x < 0 || index.y < 0 || index.x >= imageDim.x || index.y >= imageDim.y) {
        return make_uchar4(255, 255, 255, 255);
    }
    uchar4 color;
    surf2Dread(&color, image, index.x * sizeof(uchar4), index.y, hipBoundaryModeClamp);
    return color;
}

__device__ uchar4* getPixelColors(hipSurfaceObject_t image, int2 dims, int2 imageIndex, float* kernel, int kDim, int count) {
    if (kDim % 2 == 0) return nullptr;

    int half = kDim / 2;
    auto colors = new uchar4[count != 0 ? count : kDim * kDim];
    int colIndex = 0;
    for(int i=0; i < kDim; i++) {
        for(int j=0; j < kDim; j++) {
            int index = i * kDim + j;
            if (kernel[index] != 0.0) {
                int xIndex = (imageIndex.x - half + j) % dims.x;
                int yIndex = (imageIndex.y - half + i) % dims.y;
                colors[colIndex++] = getColorAt(image, dims, make_int2(xIndex, yIndex));
            }
        }
    }
    return colors;
}

__global__ void kernel_denoise(hipSurfaceObject_t image, int width, int height, int startRowIndex, int startColIndex, int sampleIndex) {
    int x = startColIndex + (int)threadIdx.x;
    int y = startRowIndex + (int)blockIdx.x;
    //uchar4 colors[5];
    //if (x > 0 && x < width-1 && y > 0 && y < height-1) {
    hiprandState state;
    hiprand_init (x*y*sampleIndex, 0, 0, &state);
    int numPixelsToSample = 5;
    float kernel[] = {0.0f, hiprand_normal(&state), 0.0f,
                      hiprand_normal(&state), hiprand_normal(&state), hiprand_normal(&state),
                      0.0f, hiprand_normal(&state), 0.0f };

//    float kernel[] = {0.0f, 0.0f, hiprand_normal(&state), 0.0f, 0.0f,
//                      0.0f, 0.0f, hiprand_normal(&state), 0.0f, 0.0f,
//                      hiprand_normal(&state), hiprand_normal(&state), hiprand_normal(&state), hiprand_normal(&state), hiprand_normal(&state),
//                      0.0f, 0.0f, hiprand_normal(&state), 0.0f, 0.0f,
//                      0.0f, 0.0f, hiprand_normal(&state), 0.0f, 0.0f};
    uchar4* colors = getPixelColors(image, make_int2(width, height), make_int2(x, y), kernel, 3, numPixelsToSample);

    if (colors != nullptr) {
        uchar4 color = colors[0];
        for (int i=1; i<numPixelsToSample; i++) {
            color.x = (color.x + colors[i].x)/2.0f;
            color.y = (color.y + colors[i].y)/2.0f;
            color.z = (color.z + colors[i].z)/2.0f;
        }
        surf2Dwrite(color, image, x * sizeof(uchar4), y, hipBoundaryModeClamp);
        free(colors);
    }
}

//----------------------------------------------------------------------------------------------------------------------
//---------------------------------------------Cuda Utils Class Definition----------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

CudaKernelUtils::CudaKernelUtils() {}
CudaKernelUtils::~CudaKernelUtils() {}

void CudaKernelUtils::initializeRenderSurface(Texture* texture) {
    size_t stackLimit = 4096;
    hipDeviceSetLimit(hipLimitStackSize, stackLimit);
    size_t newHeapLimit = 33554432;
    hipDeviceSetLimit(hipLimitMallocHeapSize, newHeapLimit);

    struct hipGraphicsResource *vbo_res;
    // register this texture with CUDA
    check(hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    check(hipGraphicsMapResources(1, &vbo_res));

    hipArray_t viewCudaArray;
    check(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, vbo_res, 0, 0));

    hipResourceDesc viewCudaArrayResourceDesc;
    memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
    viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
    viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

    check(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));
}

void CudaKernelUtils::renderScene(CudaScene* cudaScene, int blockSize, int numThreads, int startRowIndex, int startColIndex, int sampleIndex) {
    kernel_traceRays<<<blockSize, numThreads>>>(CudaKernelUtils::viewCudaSurfaceObject, cudaScene, startRowIndex, startColIndex, sampleIndex);
    check(hipDeviceSynchronize());
    //test hits
//    Bounds* test = new Bounds(0.5, -0.5, -0.5, 0.5, -2.0, -3.0);
//    std::cout << "AABB HIT: " << checkHitOnAABB(make_float3(0.0, 0.0, 0.0), make_float3(0.0, 0.0, -1.0), test) << std::endl;
}

void CudaKernelUtils::runDenoiseKernel(CudaScene* cudaScene, int blockSize, int numThreads, int startRowIndex, int startColIndex, int sampleIndex) {
    kernel_denoise<<<blockSize, numThreads>>>(CudaKernelUtils::viewCudaSurfaceObject, cudaScene->width, cudaScene->height, startRowIndex, startColIndex, sampleIndex);
    check(hipDeviceSynchronize());
}

void CudaKernelUtils::onClick(int x, int y, CudaScene* cudaScene) {
    kernel_traceSingleRay<<<1, 1>>>(CudaKernelUtils::viewCudaSurfaceObject, x, y, cudaScene);
    check(hipDeviceSynchronize());
}

void CudaKernelUtils::deviceInformation() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Thread Blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;

        hipDevice_t device;
        hipDeviceGet(&device, i);
        int major, minor;
        //hipDeviceComputeCapability(&major, &minor, device);
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);

        std::cout << "Minor: " << minor << " \nMajor: " << major << std::endl;

        size_t stackLimit;
        hipDeviceGetLimit(&stackLimit, hipLimitStackSize);
        std::cout << "Stack Size: " << stackLimit << std::endl;

        size_t heapLimit;
        hipDeviceGetLimit(&heapLimit, hipLimitMallocHeapSize);
        std::cout << "Heap Size: " << heapLimit << std::endl;
    }
}

