#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include "headers/CudaKernelUtils.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <vector_functions.h>

//----------OPERATORS---------------------------------------------------------------------------------------------------

__device__ __host__ uchar4 operator+(const uchar4 &a, const uchar4 &b) {
    return make_uchar4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float &b) {
    return make_float3(a.x*b, a.y*b, a.z*b);
}

__device__ __host__ float3 operator*(const float &a, const float3 &b) {
    return b * a;
}

__device__ __host__ float3 operator/(const float3 &a, const float &b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

__device__ __host__ float3 operator/(const float a, const float3 &b) {
    return make_float3(a/b.x, a/b.y, a/b.z);
}

__device__ __host__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ __host__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ __host__ float3 operator-(const float a, const float3 &b) {
    return make_float3(a-b.x, a-b.y, a-b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float3 &b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

//----------VECTOR--OPERATIONS------------------------------------------------------------------------------------------

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ uchar4 toRGBA(const float3 &a) {
    return make_uchar4(int(a.x * 255), int(a.y * 255), int(a.z * 255), 255);
}

__device__ __host__ float3 t_to_vec(float3 e, float3 d, float t) {
    return e + (t * d);
}

__device__ float magnitude(float3 a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ float len_squared(float3 a) {
    return a.x * a.x + a.y * a.y + a.z * a.z;
}

__device__ float3 normalize(float3 a) {
    float mag = magnitude(a);
    return make_float3(a.x, a.y, a.z)/mag;
}

__device__ float3 cross(float3 a, float3 b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

__device__ float3 clamp(float3 a, float min, float max) {
    float x = a.x; float y = a.y; float z = a.z;
    x = x < min ? min : x; x = x > max ? max : x;
    y = y < min ? min : y; y = y > max ? max : y;
    z = z < min ? min : z; z = z > max ? max : z;
    return make_float3(x, y, z);
}

//----------RT-FUNCTIONS------------------------------------------------------------------------------------------------

__device__ float3 cast_ray(unsigned int x, unsigned int y, int width, int height) {
    float d = 1.0;
    float fov = 60.0;
    float aspect_ratio = ((float)width) / ((float)height);
    float h = d * (float)tan((3.1415 * fov) / 180.0 / 2.0);
    float w = h * aspect_ratio;

    float top = h;
    float bottom = -h;
    float left = -w;
    float right = w;

    float u = left + (right - left) * float(x) / ((float)width);
    float v = bottom + (top - bottom) * (((float)height) - float(y)) / ((float)height);
    return make_float3(u, v, -d);
}

__device__ float3 getReflectedRay(float3 e, float3 d, float3 normal) {
    float3 ray_dir = normalize(d);
    return ray_dir - 2.0f * normal * dot(ray_dir, normal);
}

__device__ float3 getSphereNormal(float3 point, CudaSphere* sphere) {
    float3 normal = point - sphere->position;
    return normalize(normal);
}

__device__ float3 getTriangleNormal(float3 a, float3 b, float3 c) {
    return normalize(cross(b - a, c - a));
}

__device__ float checkHitOnPlane(float3 e, float3 d, float3 center, float3 normal) {
    /*Checks the hit on an infinite plane for the given normal and returns t value*/
    float denominator = dot(normal, d);
    if (denominator != 0.0) {
        float t = dot(normal, (center - e)) / denominator;
        return t;
    }
    return MAX_T;
}

__device__ float checkHitOnTriangle(float3 e, float3 d, float3 a, float3 b, float3 c) {
    /*Checks the hit on the triangle and returns t value. I first use the plane hit and then check if its inside the triangle*/
    float3 normal = getTriangleNormal(a, b, c);
    float t = checkHitOnPlane(e, d, a, normal);
    float3 x = t_to_vec(e, d, t);
    float aTest = dot(cross(b - a, x - a), normal);
    float bTest = dot(cross(c - b, x - b), normal);
    float cTest = dot(cross(a - c, x - c), normal);
    if (t != MAX_T && ((aTest >= 0 - HIT_T_OFFSET_1 && bTest >= 0 - HIT_T_OFFSET_1 && cTest >= 0 - HIT_T_OFFSET_1)
    || (aTest <= 0 + HIT_T_OFFSET_1 && bTest <= 0 + HIT_T_OFFSET_1 && cTest <= 0 + HIT_T_OFFSET_1))) {
        return t;
    }
    return MAX_T;
}

__device__ void printBounds(Bounds* bounds) {
    printf("AABB (%0.2f, %0.2f, %0.2f, %0.2f, %0.2f, %0.2f)", bounds->top, bounds->bottom,
           bounds->left, bounds->right, bounds->front, bounds->back);
}

__device__ void print2DUtil(BVHBinaryNode *root, int space)
{
    // Base case
    if (root == nullptr)
        return;

    // Increase distance between levels
    space += 10;

    // Process right child first
    print2DUtil(root->right, space);

    // Print current node after space
    // count
    printf("\n");
    for (int i = 10; i < space; i++)
        printf(" ");
    //print data
    printf("[{");
    //if (root->numObjects == 0) {
    //printBounds(root->bounds);
    //}
    printf("} ");
    for (int i=0; i<root->numObjects; i++) {
        printf("%d, ", root->objectsIndex[i]);
    }
    printf("]");
    //cout<<root->data<<"\n";

    // Process left child
    print2DUtil(root->left, space);
}

__device__ void swap(float &a, float &b) {
    float t = a;
    a = b;
    b = t;
}

/*
 * float3 invDir = 1.0 / d;
    //float3 invDir = make_float3(0.0, 0.0, 0.0) - d;

    float xVal = (invDir.x < 0) ? bounds->right : bounds->left;
    float yVal = (invDir.y < 0) ? bounds->top : bounds->bottom;
    float zVal = (invDir.z < 0) ? bounds->front : bounds->back;
    float tmin, tmax, tymin, tymax, tzmin, tzmax;

    tmin = (xVal - e.x) * invDir.x;
    tmax = (xVal - e.x) * invDir.x;
    tymin = (yVal - e.y) * invDir.y;
    tymax = (yVal - e.y) * invDir.y;

    if ((tmin > tymax) || (tymin > tmax))
        return MAX_T;
    if (tymin > tmin)
        tmin = tymin;
    if (tymax < tmax)
        tmax = tymax;

    tzmin = (zVal - e.z) * invDir.z;
    tzmax = (zVal - e.z) * invDir.z;

    if ((tmin > tzmax) || (tzmin > tmax))
        return MAX_T;
    if (tzmin > tmin)
        tmin = tzmin;
    if (tzmax < tmax)
        tmax = tzmax;
 */

__device__ MinMaxT checkHitOnAABB(float3 &eye, float3 &ray, Bounds* bounds, bool debug) {
    if (bounds == nullptr) return {MIN_T, MAX_T};

    float tmin = (bounds->left - eye.x) / ray.x;
    float tmax = (bounds->right - eye.x) / ray.x;

    if (tmin > tmax) swap(tmin, tmax);

    float tymin = (bounds->bottom - eye.y) / ray.y;
    float tymax = (bounds->top - eye.y) / ray.y;

    if (tymin > tymax) swap(tymin, tymax);

    if ((tmin > tymax) || (tymin > tmax))
        return {MIN_T, MAX_T};

    if (tymin > tmin)
        tmin = tymin;

    if (tymax < tmax)
        tmax = tymax;

    float tzmin = (bounds->back - eye.z) / ray.z;
    float tzmax = (bounds->front - eye.z) / ray.z;

    if (tzmin > tzmax) swap(tzmin, tzmax);

    if ((tmin > tzmax) || (tzmin > tmax))
        return {MIN_T, MAX_T};

    if (tzmin > tmin)
        tmin = tzmin;

    if (tzmax < tmax)
        tmax = tzmax;

    return {tmin != tmin ? 0 : tmin, tmax != tmax ? 0 : tmax};
}

__device__ void checkHitOnNodeTriangles(float3 &eye, float3 &ray, BVHBinaryNode* node, CudaMesh* mesh, HitInfo &hitInfo, bool debug) {
//    if (debug) {
//        printf("Starting to check hits on triangles in node NumObjects(%d)\n", node->numObjects);
//    }

    if (node != nullptr && node->numObjects != 0) { // Is a leaf node
        for (int j=0; j<node->numObjects; j++) {
            int objIndex = node->objectsIndex[j];
//            if (debug) {
//                printf("Current triangle index (%d)\n", objIndex);
//            }
            CudaTriangle t = mesh->triangles[objIndex];
            float triangleHit = checkHitOnTriangle(eye, ray, t.a, t.b, t.c);
//            if(debug) {
//                printf("Checking hits on triangle (%d) -- (%f)\n", node->objectsIndex[j], triangleHit);
//            }
            if (triangleHit <= hitInfo.t && triangleHit >= HIT_T_OFFSET_1) {
                hitInfo.t = triangleHit;
                hitInfo.hitNormal = getTriangleNormal(t.a, t.b, t.c);
//                if (debug) {
//                    printf("New hit on triangle at (%d) MinT (%f)\n", node->objectsIndex[j], hitInfo.t);
//                }
            }
        }
    }
//    if (debug) {
//        printf("Finished checking hits on triangles in node NumObjects(%d)\n", node->numObjects);
//    }
}

__device__ HitInfo checkHitOnMeshHelperNR(float3 &eye, float3 &ray, CudaMesh* mesh, bool debug) {
    if (debug) {
        printf("Starting to check hit on Mesh\n");
    }
    HitInfo hitInfo;

    auto stack = (Stack<BVHBinaryNode*>*)malloc(sizeof(Stack<BVHBinaryNode*>));
    if (!stack->init()) return hitInfo;
//    if (debug) {
//        printf("Stack initialized\n");
//    }

    float minAABB = MAX_T;
    // start from the root node (set current node to the root node)
    BVHBinaryNode* curr = mesh->bvhRoot;
//    if (debug) {
//        printf("Curr set IsNull(%d)\n", curr == nullptr);
//        printf("Stack Empty (%d)\n", stack->empty());
//    }
    // if the current node is null and the stack is also empty, we are done
    while (!stack->empty() || curr != nullptr)
    {
        // if the current node exists, push it into the stack (defer it)
        // and move to its left child
        if (curr != nullptr)
        {
//            if (debug) {
//                printf("Curr Bounds IsNull(%d)\n", curr->bounds == nullptr);
//            }
            auto currT = checkHitOnAABB(eye, ray, curr->bounds, debug);
            if (debug) {
                printf("AABB Checking curr MinT(%f) MaxT(%f) HitT(%f)\n", currT.minT, currT.maxT, hitInfo.t);
                printBounds(curr->bounds);
                printf("\n");
            }

            if (currT.minT != MIN_T && currT.maxT != MAX_T
            && currT.minT <= hitInfo.t && currT.maxT >= 0) {
                //minAABB = currT;
                checkHitOnNodeTriangles(eye, ray, curr, mesh, hitInfo, debug);
                stack->push(curr);
                curr = curr->left;
            } else {
                curr = nullptr;
                if (debug) { printf("Setting curr to null StackEmpty(%d)\n", stack->empty()); }
            }
        }
        else {
            if (debug) { printf("Curr is null | Popping off of the stack | StackEmpty(%d)\n", stack->empty()); }
            curr = stack->top();
            stack->pop();

            curr = curr->right;
        }
    }

    stack->clean();
    free(stack);
    return hitInfo;
}

__device__ float doTChecks(float newT, float oldT) {
    return newT > 0 && newT < oldT ? newT : oldT;
}

__device__ float check_hit_on_sphere(float3 &eye, float3 &ray, CudaSphere* sphere, bool debug) {
    float3 center_2_eye = eye - sphere->position;
    float ray_dot_ray = dot(ray, ray);
    float discriminant = pow(dot(ray, center_2_eye), 2) - ray_dot_ray * (dot(center_2_eye, center_2_eye) - pow(sphere->radius, 2));

    if (discriminant > 0) {
        discriminant = sqrt(discriminant);
        float init = -dot(ray, center_2_eye);
        float t1 = (init + discriminant) / ray_dot_ray;
        float t2 = (init - discriminant) / ray_dot_ray;

        float mint = min(t1, t2);
        if (mint < HIT_T_OFFSET) {
            return max(t1, t2);
        }
        return mint;
    }
    else if (discriminant == 0) {
        float init = -dot(ray, center_2_eye);
        float t1 = init / ray_dot_ray;
        return t1;
    }
    return MAX_T;
}

__device__ int wang_hash(int seed) {
    seed = (seed ^ 61) ^ (seed >> 16);
    seed *= 9;
    seed = seed ^ (seed >> 4);
    seed *= 0x27d4eb2d;
    seed = seed ^ (seed >> 15);
    return seed;
}

__device__ float newRandomFloat(CudaRandomGenerator* generator, int &randIndex) {
    if (randIndex < generator->numRand) {
        return generator->randomNumbers[randIndex++];
    } else {
        randIndex = 0;
        return generator->randomNumbers[randIndex++];
    }
}

__device__ void randFloat3(hiprandState &randState, float3 &vec) {
    vec.x = hiprand_normal(&randState);
    vec.y = hiprand_normal(&randState);
    vec.z = hiprand_normal(&randState);

    vec.x = vec.x > 1.0 ? vec.x - 1.0f : vec.x;
    vec.y = vec.y > 1.0 ? vec.y - 1.0f : vec.y;
    vec.z = vec.z > 1.0 ? vec.z - 1.0f : vec.z;
    vec.x = vec.x < -1.0 ? vec.x + 1.0f : vec.x;
    vec.y = vec.y < -1.0 ? vec.y + 1.0f : vec.y;
    vec.z = vec.z < -1.0 ? vec.z + 1.0f : vec.z;
}

__device__ float3 getNewDiffuseRay(HitInfo &hitInfo, CudaThreadData &threadData) {
    CudaRandomGenerator* generator = threadData.scene->generator;
    float3 p_n =  hitInfo.hitPoint + hitInfo.hitNormal;
//    float3 p = p_n + clamp(normalize(make_float3(newRandomFloat(generator, threadData.randIndex), newRandomFloat(generator, threadData.randIndex),
//                                 newRandomFloat(generator, threadData.randIndex))), -hitInfo.object->material->roughness, hitInfo.object->material->roughness);
    float3 vec = make_float3(0, 0, 0);
    while(true) {
        randFloat3(threadData.randState, vec);
        if (len_squared(vec) <= hitInfo.object->material->roughness) break;
    }
    float3 p = p_n + vec;
    return normalize(p - hitInfo.hitPoint);
}

__device__ float3 ray_color(const float3& r) {
    float t = 0.5f * (r.y + 1.0f);
    return (1.0f - t) * make_float3(0.5, 0.7, 1.0) + t * make_float3(1.0, 1.0, 1.0);
}

__device__ HitInfo doHitTest(float3 &eye, float3 &ray, CudaThreadData &threadData) {
    HitInfo hit;
    CudaScene* scene = threadData.scene;
    for (int i=0; i<scene->numObjects; i++) {
        if (scene->objects[i]->type == SPHERE) {
            auto sphere = (CudaSphere*)scene->objects[i];
            float sphereHit = check_hit_on_sphere(eye, ray, sphere, threadData.debug);
            if (sphereHit >= HIT_T_OFFSET && sphereHit < hit.t) {
                hit.object = sphere;
                hit.t = sphereHit;
                hit.hitPoint = t_to_vec(eye, ray, sphereHit);
                hit.hitNormal = getSphereNormal(hit.hitPoint, sphere);
                hit.color = sphere->material->diffuse;
                hit.index = i;
                if (threadData.debug) {
                    printf("doHitTest @ index (%d) with t (%f)\n", i, sphereHit);
                }
            }
        }
        else if (scene->objects[i]->type == MESH) {
            auto mesh = (CudaMesh*)scene->objects[i];
            HitInfo meshHit = checkHitOnMeshHelperNR(eye, ray, mesh, threadData.debug);
            if (meshHit.t >= HIT_T_OFFSET && meshHit.t < hit.t) {
                hit.object = mesh;
                hit.t = meshHit.t;
                hit.hitPoint = t_to_vec(eye, ray, meshHit.t);
                hit.hitNormal = meshHit.hitNormal;
                hit.color = mesh->material->diffuse;
                hit.index = i;
                if (threadData.debug) {
                    printf("doHitTest @ index (%d) with t (%f)\n", i, meshHit.t);
                }
            }
//            for (int j=0; j<mesh->numTriangles; j++) {
//                CudaTriangle t = mesh->triangles[j];
//                float triangleHit = checkHitOnTriangle(eye, ray, t.a, t.b, t.c);
//                if (triangleHit >= HIT_T_OFFSET && triangleHit < hit.t) {
//                    hit.object = mesh;
//                    hit.t = triangleHit;
//                    hit.hitPoint = t_to_vec(eye, ray, triangleHit);
//                    hit.hitNormal = getTriangleNormal(t.a, t.b, t.c);
//                    hit.index = i;
//                }
//            }
        }
    }

    for (int i=0; i<scene->numLights; i++) {
        auto light = (CudaLight*)scene->lights[i];
        if (light->lightType == SKYBOX_LIGHT) {
            auto sphere = ((CudaSkyboxLight*)scene->lights[i])->sphere;
            float sphereHit = check_hit_on_sphere(eye, ray, sphere, threadData.debug);
            if (sphereHit >= HIT_T_OFFSET && sphereHit < hit.t) {
                hit.object = light;
                hit.t = sphereHit;
                hit.hitPoint = t_to_vec(eye, ray, sphereHit);
                hit.hitNormal = getSphereNormal(hit.hitPoint, sphere);
                hit.color = ray_color(ray);
                hit.index = i;
                if (threadData.debug) {
                    printf("doHitTest @ index (%d) with t (%f)\n", i, sphereHit);
                }
            }
        }
    }
    return hit;
}

//__device__ HitInfo bounceDiffuseRays(HitInfo &hitInfo, CudaScene* scene, int &randIndex, int bounceIndex) {
//    HitInfo newHit;
//    float3 col = hitInfo.object->material->diffuse;
//    newHit.color = make_float3(col.x, col.y, col.z);
//    int numRaySamples = 4;
//    for(int n=0; n<numRaySamples; n++) {
//        float3 reflected = getNewDiffuseRay(scene, hitInfo, randIndex);
//        HitInfo t = doHitTest(hitInfo.hitPoint, reflected, scene, false);
//        if (t.object->type == LIGHT && ((CudaLight*)t.object)->lightType == SKYBOX_LIGHT) {
//            newHit.color = newHit.color + 0.5 * ray_color(reflected);
//        }
//    }
//    newHit.color = (newHit.color/(float)numRaySamples);
//    return newHit;
//}

__device__ float3 calculateLighting(HitInfo &hitInfo, CudaThreadData &threadData) {
    if (hitInfo.object->type == LIGHT) {
        return hitInfo.color;
    }
    CudaScene* scene = threadData.scene;
    float3 lighting = make_float3(0, 0, 0);
    for (int i=0; i<scene->numLights; i++) {
        auto light = (CudaLight*)scene->lights[i];
        if (light->lightType == SKYBOX_LIGHT) {
            float3 col = hitInfo.object->material->diffuse;
            float3 tempColor = make_float3(col.x, col.y, col.z);
            int numRaySamples = 8;
            for(int n=0; n<numRaySamples; n++) {
                float3 reflected = getNewDiffuseRay(hitInfo, threadData);
                HitInfo newHit = doHitTest(hitInfo.hitPoint, reflected, threadData);
                if (newHit.object->type == LIGHT && newHit.index == i) {
                    tempColor = tempColor + 0.5 * ray_color(reflected);
                }
            }
            lighting = lighting + (tempColor/(float)numRaySamples);
        }
    }
    return lighting;
}

__device__ float3 traceSingleRay(float3 eye, float3 ray, int maxBounces, CudaThreadData &threadData) {
    auto stack = (Stack<HitInfo>*)malloc(sizeof(Stack<HitInfo>));
    stack->init();

    float3 newRay = ray;
    float3 newEye = eye;
    bool isHit = true;
    int bounceIndex = 0;
    while(bounceIndex < maxBounces && isHit) {
        HitInfo hitInfo = doHitTest(newEye, newRay, threadData);
        if (hitInfo.isHit()) {
            stack->push(hitInfo);
            if (hitInfo.object->type >= LIGHT) {
                // skybox obj
                isHit = false;
                break;
            }

            //newRay = normalize(getReflectedRay(eye, ray, hitInfo.hitNormal));
            newRay = getNewDiffuseRay(hitInfo, threadData);
            newEye = hitInfo.hitPoint;
            if (threadData.debug) {
                printf("HitInfo(%d); Hit T(%f) @ (%f, %f, %f) | Normal(%f, %f, %f) | Reflected(%f, %f, %f)\n",
                       hitInfo.index, hitInfo.t, hitInfo.hitPoint.x, hitInfo.hitPoint.y, hitInfo.hitPoint.z,
                       hitInfo.hitNormal.x, hitInfo.hitNormal.y, hitInfo.hitNormal.z,
                       newRay.x, newRay.y, newRay.z);
            }
        } else {
            isHit = false;
        }
        bounceIndex++;
    }

    // Sum all colors from stack
    float3 color = make_float3(0.0, 0.0, 0.0);
    if (stack->size() >= 2) {
        while(!stack->empty()) {
            HitInfo curr = stack->top();
            stack->pop();
//            if (debug) {
//                printf("\n\nCurr | Index(%d)\n", curr.index);
//                printf("Next | Index(%d)\n", curr.index);
//                printf("Color(%f, %f, %f)\n", color.x, color.y, color.z);
//            }
            if (curr.object->type < LIGHT) {
                //color = curr.color + (curr.object->material->reflective * color);
                float3 ref = curr.object->material->reflective;
                float3 currLighting = calculateLighting(curr, threadData);
                color = (1.0 - ref) * currLighting + (ref * color);
            } else {
                color = curr.color;
            }
        }
    } else if (stack->size() == 1) {
        HitInfo curr = stack->top(); stack->pop();
        color = color + curr.color;
    }

    stack->clean();
    free(stack);

    return color;
}

__global__ void kernel_traceRays(hipSurfaceObject_t image, CudaScene* scene,  int startRowIndex, int startColIndex)
{
    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    int x = startColIndex + (int)threadIdx.x;
    int y = startRowIndex + (int)blockIdx.x;
    int randIndex = wang_hash(x * y) % scene->width;

    hiprandState state;
    hiprand_init (x*y, 0, 0, &state);

    CudaThreadData threadData;
    threadData.debug = false;
    threadData.randState = state;
    threadData.randIndex = randIndex;
    threadData.scene = scene;

    int maxBounces = 4;
    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, scene->width, scene->height) - eye;

    int numSamples = 4;
    float3 sampledColor = make_float3(0, 0, 0);
    float p = 0.0005;
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(p, p, 0), maxBounces, threadData);
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(-p, -p, 0), maxBounces, threadData);
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(-p, p, 0), maxBounces, threadData);
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(p, -p, 0), maxBounces, threadData);
    uchar4 color = toRGBA(sampledColor/(float)numSamples);

//    float3 cVal = traceSingleRay(eye, ray, scene, maxBounces, randIndex, false);
//    uchar4 color = toRGBA(cVal);

    surf2Dwrite(color, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
}

__global__ void kernel_traceSingleRay(hipSurfaceObject_t image, int x, int y, CudaScene* scene)
{

//    printf("New Rand %f\n", hiprand_normal(&s));
//    printf("New Rand %f\n", hiprand_normal(&s));
//    printf("New Rand %f\n", hiprand_normal(&s));

    int randIndex = (x * y) % scene->width;
    int maxBounces = 8;

    CudaThreadData threadData;
    hiprandState state;
    hiprand_init (x*y, 0, 0, &state);
    threadData.debug = true;
    threadData.randState = state;
    threadData.randIndex = randIndex;
    threadData.scene = scene;

//    for (int i=0; i<20; i++) {
//        float3 vec = randFloat3(threadData.randState);
//        vec.x = vec.x > 1.0 ? vec.x - 1.0f : vec.x;
//        vec.y = vec.y > 1.0 ? vec.y - 1.0f : vec.y;
//        vec.z = vec.z > 1.0 ? vec.z - 1.0f : vec.z;
//
//        vec.x = vec.x < -1.0 ? vec.x + 1.0f : vec.x;
//        vec.y = vec.y < -1.0 ? vec.y + 1.0f : vec.y;
//        vec.z = vec.z < -1.0 ? vec.z + 1.0f : vec.z;
//
//        printf("New Vector (%f, %f, %f)\n", vec.x, vec.y, vec.z);
//        printf("Len Squared %f\n", len_squared(vec));
//    }

//    float3 eye = make_float3(0.0, 0.0, 0.0);
//    float3 ray = cast_ray(x, y, scene->width, scene->height) - eye;
//    printf("\n\nRay (%f, %f, %f)\n", ray.x, ray.y, ray.z);
//    uchar4 color = toRGBA(traceSingleRay(eye, ray, maxBounces, threadData));
//    printf("Final Color: (%d, %d, %d, %d)\n", color.x, color.y, color.z, color.w);
//    surf2Dwrite(color, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
}

//----------------------------------------------------------------------------------------------------------------------
//---------------------------------------------Cuda Utils Class Definition----------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

CudaKernelUtils::CudaKernelUtils() {}
CudaKernelUtils::~CudaKernelUtils() {}

void CudaKernelUtils::initializeRenderSurface(Texture* texture) {
    size_t stackLimit = 2048;
    hipDeviceSetLimit(hipLimitStackSize, stackLimit);
    size_t newHeapLimit = 16777216;
    hipDeviceSetLimit(hipLimitMallocHeapSize, newHeapLimit);
//    size_t getLimit;
//    hipDeviceGetLimit(&getLimit, hipLimitStackSize);
//    std::cout << "New Stack Size: " << getLimit << std::endl;



    struct hipGraphicsResource *vbo_res;
    // register this texture with CUDA
    //hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(),GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);
    check(hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    check(hipGraphicsMapResources(1, &vbo_res));

    hipArray_t viewCudaArray;
    check(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, vbo_res, 0, 0));

    hipResourceDesc viewCudaArrayResourceDesc;
    memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
    viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
    viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

    check(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));

//    hiprandGenerator_t  randomGenerator;
//    hiprandCreateGenerator(&randomGenerator, HIPRAND_RNG_QUASI_SOBOL32);
//    hiprandSetPseudoRandomGeneratorSeed(randomGenerator, 1);

}

void CudaKernelUtils::renderScene(CudaScene* cudaScene, int blockSize, int numThreads, int startRowIndex, int startColIndex) {
    kernel_traceRays<<<blockSize, numThreads>>>(CudaKernelUtils::viewCudaSurfaceObject, cudaScene, startRowIndex, startColIndex);
    check(hipDeviceSynchronize());
    //test hits
//    Bounds* test = new Bounds(0.5, -0.5, -0.5, 0.5, -2.0, -3.0);
//    std::cout << "AABB HIT: " << checkHitOnAABB(make_float3(0.0, 0.0, 0.0), make_float3(0.0, 0.0, -1.0), test) << std::endl;
}

void CudaKernelUtils::onClick(int x, int y, CudaScene* cudaScene) {
    kernel_traceSingleRay<<<1, 1>>>(CudaKernelUtils::viewCudaSurfaceObject, x, y, cudaScene);
    check(hipDeviceSynchronize());
}

void CudaKernelUtils::deviceInformation() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Thread Blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;

        hipDevice_t device;
        hipDeviceGet(&device, i);
        int major, minor;
        //hipDeviceComputeCapability(&major, &minor, device);
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);

        std::cout << "Minor: " << minor << " \nMajor: " << major << std::endl;

        size_t stackLimit;
        hipDeviceGetLimit(&stackLimit, hipLimitStackSize);
        std::cout << "Stack Size: " << stackLimit << std::endl;

        size_t heapLimit;
        hipDeviceGetLimit(&heapLimit, hipLimitMallocHeapSize);
        std::cout << "Heap Size: " << heapLimit << std::endl;
    }
}

