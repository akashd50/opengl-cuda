#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include "headers/CudaKernelUtils.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <vector_functions.h>

//----------OPERATORS---------------------------------------------------------------------------------------------------

__device__ __host__ uchar4 operator+(const uchar4 &a, const uchar4 &b) {
    return make_uchar4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float &b) {
    return make_float3(a.x*b, a.y*b, a.z*b);
}

__device__ __host__ float3 operator*(const float &a, const float3 &b) {
    return b * a;
}

__device__ __host__ float3 operator/(const float3 &a, const float &b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

__device__ __host__ float3 operator/(const float a, const float3 &b) {
    return make_float3(a/b.x, a/b.y, a/b.z);
}

__device__ __host__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ __host__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ __host__ float3 operator-(const float a, const float3 &b) {
    return make_float3(a-b.x, a-b.y, a-b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float3 &b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

//----------VECTOR--OPERATIONS------------------------------------------------------------------------------------------

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ uchar4 toRGBA(const float3 &a) {
    return make_uchar4(int(a.x * 255), int(a.y * 255), int(a.z * 255), 255);
}

__device__ __host__ float3 t_to_vec(float3 e, float3 d, float t) {
    return e + (t * d);
}

__device__ float magnitude(float3 a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ float len_squared(float3 a) {
    return a.x * a.x + a.y * a.y + a.z * a.z;
}

__device__ float3 normalize(float3 a) {
    float mag = magnitude(a);
    return make_float3(a.x, a.y, a.z)/mag;
}

__device__ float3 cross(float3 a, float3 b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

__device__ float3 clamp(float3 a, float min, float max) {
    float x = a.x; float y = a.y; float z = a.z;
    x = x < min ? min : x; x = x > max ? max : x;
    y = y < min ? min : y; y = y > max ? max : y;
    z = z < min ? min : z; z = z > max ? max : z;
    return make_float3(x, y, z);
}

//----------RT-FUNCTIONS------------------------------------------------------------------------------------------------

__device__ float3 cast_ray(unsigned int x, unsigned int y, int width, int height) {
    float d = 1.0;
    float fov = 60.0;
    float aspect_ratio = ((float)width) / ((float)height);
    float h = d * (float)tan((3.1415 * fov) / 180.0 / 2.0);
    float w = h * aspect_ratio;

    float top = h;
    float bottom = -h;
    float left = -w;
    float right = w;

    float u = left + (right - left) * float(x) / ((float)width);
    float v = bottom + (top - bottom) * (((float)height) - float(y)) / ((float)height);
    return make_float3(u, v, -d);
}

__device__ float3 getReflectedRay(float3 e, float3 d, float3 normal) {
    float3 ray_dir = normalize(d);
    return ray_dir - 2.0f * normal * dot(ray_dir, normal);
}

__device__ float3 getSphereNormal(float3 point, CudaSphere* sphere) {
    float3 normal = point - sphere->position;
    return normalize(normal);
}

__device__ float3 getTriangleNormal(float3 a, float3 b, float3 c) {
    return normalize(cross(b - a, c - a));
}

__device__ float checkHitOnPlane(float3 e, float3 d, float3 center, float3 normal) {
    /*Checks the hit on an infinite plane for the given normal and returns t value*/
    float denominator = dot(normal, d);
    if (denominator != 0.0) {
        float t = dot(normal, (center - e)) / denominator;
        return t;
    }
    return MAX_T;
}

__device__ float checkHitOnTriangle(float3 e, float3 d, float3 a, float3 b, float3 c) {
    /*Checks the hit on the triangle and returns t value. I first use the plane hit and then check if its inside the triangle*/
    float3 normal = getTriangleNormal(a, b, c);
    float t = checkHitOnPlane(e, d, a, normal);
    float3 x = t_to_vec(e, d, t);
    float aTest = dot(cross(b - a, x - a), normal);
    float bTest = dot(cross(c - b, x - b), normal);
    float cTest = dot(cross(a - c, x - c), normal);
    if (t != MAX_T && ((aTest >= 0 - HIT_T_OFFSET_1 && bTest >= 0 - HIT_T_OFFSET_1 && cTest >= 0 - HIT_T_OFFSET_1)
    || (aTest <= 0 + HIT_T_OFFSET_1 && bTest <= 0 + HIT_T_OFFSET_1 && cTest <= 0 + HIT_T_OFFSET_1))) {
        return t;
    }
    return MAX_T;
}

__device__ bool check_hit_on_sphere(float3 &eye, float3 &ray, CudaSphere* sphere, HitInfo &hitInfo, bool debug) {
    float3 center_2_eye = eye - sphere->position;
    float ray_dot_ray = dot(ray, ray);
    float discriminant = pow(dot(ray, center_2_eye), 2) - ray_dot_ray * (dot(center_2_eye, center_2_eye) - pow(sphere->radius, 2));
    float mint = MAX_T;
    if (discriminant > 0) {
        discriminant = sqrt(discriminant);
        float init = -dot(ray, center_2_eye);
        float t1 = (init + discriminant) / ray_dot_ray;
        float t2 = (init - discriminant) / ray_dot_ray;

        mint = min(t1, t2);
        if (mint < HIT_T_OFFSET) {
            mint = max(t1, t2);
        }
    }
    else if (discriminant == 0) {
        float init = -dot(ray, center_2_eye);
        mint = init / ray_dot_ray;
    }

    if (mint >= HIT_T_OFFSET && mint < hitInfo.t) {
        hitInfo.t = mint;
        return true;
    }
    return false;
}

__device__ void printBounds(Bounds* bounds) {
    printf("AABB (%0.2f, %0.2f, %0.2f, %0.2f, %0.2f, %0.2f)", bounds->top, bounds->bottom,
           bounds->left, bounds->right, bounds->front, bounds->back);
}

__device__ void print2DUtil(BVHBinaryNode *root, int space)
{
    // Base case
    if (root == nullptr)
        return;

    // Increase distance between levels
    space += 10;

    // Process right child first
    print2DUtil(root->right, space);

    // Print current node after space
    // count
    printf("\n");
    for (int i = 10; i < space; i++)
        printf(" ");
    //print data
    printf("[{");
    //if (root->numObjects == 0) {
    //printBounds(root->bounds);
    //}
    printf("} ");
    for (int i=0; i<root->numObjects; i++) {
        printf("%d, ", root->objectsIndex[i]);
    }
    printf("]");
    //cout<<root->data<<"\n";

    // Process left child
    print2DUtil(root->left, space);
}

__device__ void swap(float &a, float &b) {
    float t = a;
    a = b;
    b = t;
}

/*
 * float3 invDir = 1.0 / d;
    //float3 invDir = make_float3(0.0, 0.0, 0.0) - d;

    float xVal = (invDir.x < 0) ? bounds->right : bounds->left;
    float yVal = (invDir.y < 0) ? bounds->top : bounds->bottom;
    float zVal = (invDir.z < 0) ? bounds->front : bounds->back;
    float tmin, tmax, tymin, tymax, tzmin, tzmax;

    tmin = (xVal - e.x) * invDir.x;
    tmax = (xVal - e.x) * invDir.x;
    tymin = (yVal - e.y) * invDir.y;
    tymax = (yVal - e.y) * invDir.y;

    if ((tmin > tymax) || (tymin > tmax))
        return MAX_T;
    if (tymin > tmin)
        tmin = tymin;
    if (tymax < tmax)
        tmax = tymax;

    tzmin = (zVal - e.z) * invDir.z;
    tzmax = (zVal - e.z) * invDir.z;

    if ((tmin > tzmax) || (tzmin > tmax))
        return MAX_T;
    if (tzmin > tmin)
        tmin = tzmin;
    if (tzmax < tmax)
        tmax = tzmax;
 */

__device__ MinMaxT checkHitOnAABB(float3 &eye, float3 &ray, Bounds* bounds, bool debug) {
    if (bounds == nullptr) return {MIN_T, MAX_T};

    float tmin = (bounds->left - eye.x) / ray.x;
    float tmax = (bounds->right - eye.x) / ray.x;

    if (tmin > tmax) swap(tmin, tmax);

    float tymin = (bounds->bottom - eye.y) / ray.y;
    float tymax = (bounds->top - eye.y) / ray.y;

    if (tymin > tymax) swap(tymin, tymax);

    if ((tmin > tymax) || (tymin > tmax))
        return {MIN_T, MAX_T};

    if (tymin > tmin)
        tmin = tymin;

    if (tymax < tmax)
        tmax = tymax;

    float tzmin = (bounds->back - eye.z) / ray.z;
    float tzmax = (bounds->front - eye.z) / ray.z;

    if (tzmin > tzmax) swap(tzmin, tzmax);

    if ((tmin > tzmax) || (tzmin > tmax))
        return {MIN_T, MAX_T};

    if (tzmin > tmin)
        tmin = tzmin;

    if (tzmax < tmax)
        tmax = tzmax;

    return {tmin != tmin ? 0 : tmin, tmax != tmax ? 0 : tmax};
}

__device__ bool checkHitOnNodeTriangles(float3 &eye, float3 &ray, BVHBinaryNode* node, CudaMesh* mesh, HitInfo &hitInfo, bool debug) {
//    if (debug) {
//        printf("Starting to check hits on triangles in node NumObjects(%d)\n", node->numObjects);
//    }
    bool tUpdated = false;
    if (node != nullptr && node->numObjects != 0) { // Is a leaf node
        for (int j=0; j<node->numObjects; j++) {
            int objIndex = node->objectsIndex[j];
//            if (debug) {
//                printf("Current triangle index (%d)\n", objIndex);
//            }
            CudaTriangle t = mesh->triangles[objIndex];
            float triangleHit = checkHitOnTriangle(eye, ray, t.a, t.b, t.c);
//            if(debug) {
//                printf("Checking hits on triangle (%d) -- (%f)\n", node->objectsIndex[j], triangleHit);
//            }
            if (triangleHit < hitInfo.t && triangleHit >= HIT_T_OFFSET_1) {
                hitInfo.t = triangleHit;
                hitInfo.normal = getTriangleNormal(t.a, t.b, t.c);
                tUpdated = true;
                if (debug) {
                    printf("New hit on triangle at (%d) MinT (%f)\n", node->objectsIndex[j], hitInfo.t);
                }
            }
        }
    }

    return tUpdated;
}

__device__ bool checkHitOnMeshHelperNR(float3 &eye, float3 &ray, CudaMesh* mesh, HitInfo &hitInfo, bool debug) {
    if (debug) {
        printf("\n\nStarting to check hit on Mesh\n");
    }

    //auto stack = (Stack<BVHBinaryNode*>*)malloc(sizeof(Stack<BVHBinaryNode*>));
    auto stack = new Stack<BVHBinaryNode*>();
    if (!stack->init()) return false;
//    if (debug) {
//        printf("Stack initialized\n");
//    }

    //float minAABB = MAX_T;
    // start from the root node (set current node to the root node)
    BVHBinaryNode* curr = mesh->bvhRoot;
//    if (debug) {
//        printf("Curr set IsNull(%d)\n", curr == nullptr);
//        printf("Stack Empty (%d)\n", stack->empty());
//    }
    // if the current node is null and the stack is also empty, we are done
    bool tUpdated = false;
    while (!stack->empty() || curr != nullptr)
    {
        // if the current node exists, push it into the stack (defer it)
        // and move to its left child
        if (curr != nullptr)
        {
//            if (debug) {
//                printf("Curr Bounds IsNull(%d)\n", curr->bounds == nullptr);
//            }
            auto currT = checkHitOnAABB(eye, ray, curr->bounds, debug);
            if (debug) {
                printf("AABB Checking curr MinT(%f) MaxT(%f) HitT(%f)\n", currT.minT, currT.maxT, hitInfo.t);
                printBounds(curr->bounds);
                printf("\n");
            }

            if (currT.minT != MIN_T && currT.maxT != MAX_T
            && currT.minT <= hitInfo.t && currT.maxT >= 0) {
                //minAABB = currT;
                if (checkHitOnNodeTriangles(eye, ray, curr, mesh, hitInfo, debug)) {
                    tUpdated = true;
                }

                stack->push(curr);
                curr = curr->left;
            } else {
                curr = nullptr;
                if (debug) { printf("Setting curr to null StackEmpty(%d)\n", stack->empty()); }
            }
        }
        else {
            if (debug) { printf("Curr is null | Popping off of the stack | StackEmpty(%d)\n", stack->empty()); }
            curr = stack->top();
            stack->pop();

            curr = curr->right;
        }
    }

    stack->clean();
    free(stack);
    return tUpdated;
}

__device__ int wang_hash(int seed) {
    seed = (seed ^ 61) ^ (seed >> 16);
    seed *= 9;
    seed = seed ^ (seed >> 4);
    seed *= 0x27d4eb2d;
    seed = seed ^ (seed >> 15);
    return seed;
}

__device__ void randFloat3(hiprandState &randState, float3 &vec) {
    vec.x = hiprand_normal(&randState);
    vec.y = hiprand_normal(&randState);
    vec.z = hiprand_normal(&randState);
}

__device__ float3 getReflectedDiffuseRay(HitInfo &hitInfo, CudaThreadData &threadData, bool useReflected) {
    float3 p_n = hitInfo.point + (useReflected ? hitInfo.reflected * 1.4 : hitInfo.normal * 1.4);
    float3 vec = make_float3(0, 0, 0);
    randFloat3(threadData.randState, vec);

    /*if (len_squared(vec) > hitInfo.object->material->roughness)*/ vec = normalize(vec);
    vec = vec * hitInfo.object->material->roughness;

    float3 p = p_n + vec;
    return normalize(p - hitInfo.point);
}

__device__ float3 ray_color(const float3& r) {
    float t = 0.5f * (r.y + 1.0f);
    return (1.0f - t) * make_float3(0.5, 0.7, 1.0) + t * make_float3(1.0, 1.0, 1.0);
}

__device__ void doHitTest(float3 &eye, float3 &ray, HitInfo &hitInfo, CudaThreadData &threadData) {
    CudaScene* scene = threadData.scene;
    for (int i=0; i<scene->numObjects; i++) {
        if (scene->objects[i]->type == SPHERE) {
            auto sphere = (CudaSphere*)scene->objects[i];
            if (check_hit_on_sphere(eye, ray, sphere, hitInfo, threadData.debug)) {
                hitInfo.object = sphere;
                hitInfo.point = t_to_vec(eye, ray, hitInfo.t);
                hitInfo.normal = getSphereNormal(hitInfo.point, sphere);
                hitInfo.color = sphere->material->diffuse;
                hitInfo.index = i;
                if (threadData.debug) {
                    printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
                }
            }
        }
        else if (scene->objects[i]->type == MESH) {
            auto mesh = (CudaMesh*)scene->objects[i];
            if (checkHitOnMeshHelperNR(eye, ray, mesh, hitInfo, threadData.debug)) {
                hitInfo.object = mesh;
                hitInfo.point = t_to_vec(eye, ray, hitInfo.t);
                hitInfo.color = mesh->material->diffuse;
                hitInfo.index = i;
                if (threadData.debug) {
                    printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
                }
            }
        }
    }

    for (int i=0; i<scene->numLights; i++) {
        auto light = (CudaLight*)scene->lights[i];
        if (light->lightType == SKYBOX_LIGHT) {
            auto sphere = ((CudaSkyboxLight*)scene->lights[i])->sphere;
            if (check_hit_on_sphere(eye, ray, sphere, hitInfo, threadData.debug)) {
                hitInfo.object = light;
                hitInfo.point = t_to_vec(eye, ray, hitInfo.t);
                hitInfo.normal = getSphereNormal(hitInfo.point, sphere);
                hitInfo.color = ray_color(ray);
                hitInfo.index = i;
                if (threadData.debug) {
                    printf("\n\ndoHitTest @ index (%d) with t (%f)\n", i, hitInfo.t);
                }
            }
        }
    }
}

__device__ float3 calculateLighting(HitInfo &hitInfo, CudaThreadData &threadData) {
    if (threadData.debug) {
        printf("\n\nCalculating Lighting for Hit(%d) T(%f) P(%f, %f, %f) | Normal(%f, %f, %f)\n", hitInfo.index, hitInfo.t,
               hitInfo.point.x, hitInfo.point.y, hitInfo.point.z, hitInfo.normal.x, hitInfo.normal.y, hitInfo.normal.z);
    }

    if (hitInfo.object->type == LIGHT) {
        return hitInfo.color;
    }

    CudaScene* scene = threadData.scene;
    float3 lighting = make_float3(0, 0, 0);
    for (int i=0; i<scene->numLights; i++) {
        auto light = (CudaLight*)scene->lights[i];
        if (light->lightType == SKYBOX_LIGHT) {
            float3 diffuse = hitInfo.object->material->diffuse;
            float3 tempColor = make_float3(0, 0, 0);
            int numRaySamples = 1;
            HitInfo newHit;
            for(int n=0; n<numRaySamples; n++) {
                float3 reflected = getReflectedDiffuseRay(hitInfo, threadData, false);
                doHitTest(hitInfo.point, reflected, newHit, threadData);
                if (threadData.debug) {
                    printf("\n\nHitInfo(%d) Type(%d); Hit T(%f) @ (%f, %f, %f) | Normal(%f, %f, %f) | Ray(%f, %f, %f)\n",
                           newHit.index, newHit.object->type, newHit.t, newHit.point.x, newHit.point.y, newHit.point.z,
                           newHit.normal.x, newHit.normal.y, newHit.normal.z,
                           reflected.x, reflected.y, reflected.z);
                }
                if (newHit.object->type == LIGHT
                && newHit.index == i) {
                    tempColor = tempColor + ray_color(reflected);
                }
                else {
                    float3 newReflected = getReflectedDiffuseRay(newHit, threadData, false);
                    doHitTest(newHit.point, newReflected, newHit, threadData);
                    if (newHit.object->type == LIGHT && newHit.index == i) {
                        tempColor = tempColor + ray_color(newReflected);
                    }
                }
            }

            tempColor = (tempColor/(float)numRaySamples) * diffuse;
            //tempColor = tempColor * diffuse;
            lighting = lighting + tempColor;
        }
    }
    return lighting;
}

__device__ float3 traceSingleRay(float3 eye, float3 ray, int maxBounces, CudaThreadData &threadData) {
    //auto stack = (Stack<HitInfo>*)malloc(sizeof(Stack<HitInfo>));
    auto stack = new Stack<HitInfo>();
    stack->init();

    float3 newRay = ray;
    float3 newEye = eye;
    bool isHit = true;
    int bounceIndex = 0;
    while(bounceIndex < maxBounces && isHit) {
        HitInfo hitInfo;
        doHitTest(newEye, newRay, hitInfo, threadData);
        if (hitInfo.isHit()) {
            stack->push(hitInfo);
            if (hitInfo.object->type >= LIGHT) {
                // skybox obj
                isHit = false;
                break;
            }

            hitInfo.reflected = normalize(getReflectedRay(eye, ray, hitInfo.normal));
            newRay = getReflectedDiffuseRay(hitInfo, threadData, true); // Need to consider the incoming ray for reflectance
            newEye = hitInfo.point;
            if (threadData.debug) {
                printf("\n\nHitInfo(%d); Hit T(%f) @ (%f, %f, %f) | Normal(%f, %f, %f) | Reflected(%f, %f, %f)\n",
                       hitInfo.index, hitInfo.t, hitInfo.point.x, hitInfo.point.y, hitInfo.point.z,
                       hitInfo.normal.x, hitInfo.normal.y, hitInfo.normal.z,
                       newRay.x, newRay.y, newRay.z);
            }
        } else {
            isHit = false;
        }
        bounceIndex++;
    }

    // Sum all colors from stack
    float3 color = make_float3(0.0, 0.0, 0.0);
    if (stack->size() >= 2) {
        while(!stack->empty()) {
            HitInfo curr = stack->top();
            stack->pop();
//            if (debug) {
//                printf("\n\nCurr | Index(%d)\n", curr.index);
//                printf("Next | Index(%d)\n", curr.index);
//                printf("Color(%f, %f, %f)\n", color.x, color.y, color.z);
//            }
            if (curr.object->type < LIGHT) {
                //color = curr.color + (curr.object->material->reflective * color);
                float3 ref = curr.object->material->reflective;
                float3 currLighting = calculateLighting(curr, threadData);
                color = (1.0 - ref) * currLighting + (ref * color);
            } else {
                color = curr.color;
            }
        }
    } else if (stack->size() == 1) {
        HitInfo curr = stack->top(); stack->pop();
        color = color + curr.color;
    }

    stack->clean();
    free(stack);

    return color;
}

__global__ void kernel_traceRays(hipSurfaceObject_t image, CudaScene* scene,  int startRowIndex, int startColIndex)
{
    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    int x = startColIndex + (int)threadIdx.x;
    int y = startRowIndex + (int)blockIdx.x;
    int randIndex = wang_hash(x * y) % scene->width;

    hiprandState state;
    hiprand_init (x*y, 0, 0, &state);

    CudaThreadData threadData;
    threadData.debug = false;
    threadData.randState = state;
    threadData.randIndex = randIndex;
    threadData.scene = scene;

    int maxBounces = 4;
    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, scene->width, scene->height) - eye;

    int numSamples = 4;
    float3 sampledColor = make_float3(0, 0, 0);
    float p = 0.0005f /* hiprand_normal(&threadData.randState)*/;
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(p, p, 0), maxBounces, threadData);
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(-p, -p, 0), maxBounces, threadData);
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(-p, p, 0), maxBounces, threadData);
    sampledColor = sampledColor + traceSingleRay(eye, ray + make_float3(p, -p, 0), maxBounces, threadData);
    uchar4 color = toRGBA(sampledColor/(float)numSamples);

//    float3 cVal = traceSingleRay(eye, ray, scene, maxBounces, randIndex, false);
//    uchar4 color = toRGBA(cVal);

    surf2Dwrite(color, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
}

__global__ void kernel_traceSingleRay(hipSurfaceObject_t image, int x, int y, CudaScene* scene)
{
    int randIndex = (x * y) % scene->width;
    int maxBounces = 8;

    CudaThreadData threadData;
    hiprandState state;
    hiprand_init (x*y, 0, 0, &state);
    threadData.debug = true;
    threadData.randState = state;
    threadData.randIndex = randIndex;
    threadData.scene = scene;

    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, scene->width, scene->height) - eye;
    printf("\n\nRay (%f, %f, %f)\n", ray.x, ray.y, ray.z);
    uchar4 color = toRGBA(traceSingleRay(eye, ray, maxBounces, threadData));
    printf("Final Color: (%d, %d, %d, %d)\n", color.x, color.y, color.z, color.w);
    surf2Dwrite(color, image, x * sizeof(uchar4), scene->height - y, hipBoundaryModeClamp);
}

//----------------------------------------------------------------------------------------------------------------------
//---------------------------------------------Cuda Utils Class Definition----------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

CudaKernelUtils::CudaKernelUtils() {}
CudaKernelUtils::~CudaKernelUtils() {}

void CudaKernelUtils::initializeRenderSurface(Texture* texture) {
    size_t stackLimit = 4096;
    hipDeviceSetLimit(hipLimitStackSize, stackLimit);
    size_t newHeapLimit = 33554432;
    hipDeviceSetLimit(hipLimitMallocHeapSize, newHeapLimit);
//    size_t getLimit;
//    hipDeviceGetLimit(&getLimit, hipLimitStackSize);
//    std::cout << "New Stack Size: " << getLimit << std::endl;



    struct hipGraphicsResource *vbo_res;
    // register this texture with CUDA
    //hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(),GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);
    check(hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    check(hipGraphicsMapResources(1, &vbo_res));

    hipArray_t viewCudaArray;
    check(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, vbo_res, 0, 0));

    hipResourceDesc viewCudaArrayResourceDesc;
    memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
    viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
    viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

    check(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));

//    hiprandGenerator_t  randomGenerator;
//    hiprandCreateGenerator(&randomGenerator, HIPRAND_RNG_QUASI_SOBOL32);
//    hiprandSetPseudoRandomGeneratorSeed(randomGenerator, 1);

}

void CudaKernelUtils::renderScene(CudaScene* cudaScene, int blockSize, int numThreads, int startRowIndex, int startColIndex) {
    kernel_traceRays<<<blockSize, numThreads>>>(CudaKernelUtils::viewCudaSurfaceObject, cudaScene, startRowIndex, startColIndex);
    check(hipDeviceSynchronize());
    //test hits
//    Bounds* test = new Bounds(0.5, -0.5, -0.5, 0.5, -2.0, -3.0);
//    std::cout << "AABB HIT: " << checkHitOnAABB(make_float3(0.0, 0.0, 0.0), make_float3(0.0, 0.0, -1.0), test) << std::endl;
}

void CudaKernelUtils::onClick(int x, int y, CudaScene* cudaScene) {
    kernel_traceSingleRay<<<1, 1>>>(CudaKernelUtils::viewCudaSurfaceObject, x, y, cudaScene);
    check(hipDeviceSynchronize());
}

void CudaKernelUtils::deviceInformation() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Thread Blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;

        hipDevice_t device;
        hipDeviceGet(&device, i);
        int major, minor;
        //hipDeviceComputeCapability(&major, &minor, device);
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);

        std::cout << "Minor: " << minor << " \nMajor: " << major << std::endl;

        size_t stackLimit;
        hipDeviceGetLimit(&stackLimit, hipLimitStackSize);
        std::cout << "Stack Size: " << stackLimit << std::endl;

        size_t heapLimit;
        hipDeviceGetLimit(&heapLimit, hipLimitMallocHeapSize);
        std::cout << "Heap Size: " << heapLimit << std::endl;
    }
}

