#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include <math.h>
#include "glm/glm.hpp"
#include "headers/MainCuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <surface_functions.h>
#include <surface_indirect_functions.h>
#include <hip/hip_vector_types.h>
#include <math_functions.h>

__device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__global__
void add(int n, float *x, float *y)
{

    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    int index = threadIdx.x;
    printf("Index %d", index);
    int stride = blockDim.x;
    for (int i = index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

__device__ float3 cast_ray(unsigned int x, unsigned int y, int width, int height) {
    float d = 1.0;
    float fov = 60.0;
    float aspect_ratio = ((float)width) / ((float)height);
    float h = d * (float)tan((3.1415 * fov) / 180.0 / 2.0);
    float w = h * aspect_ratio;

    float top = h;
    float bottom = -h;
    float left = -w;
    float right = w;

    float u = left + (right - left) * float(x) / ((float)width);
    float v = bottom + (top - bottom) * (((float)height) - float(y)) / ((float)height);
    return make_float3(u, v, -d);
}

__device__ const float MIN_T = -9999.0;
__device__ const float HIT_T_OFFSET = 0.01;
//
__device__ float check_hit_on_sphere(float3 eye, float3 ray, float3 center, float radius) {
    float3 center_2_eye = eye - center;
    float ray_dot_ray = dot(ray, ray);
    float discriminant = pow(dot(ray, center_2_eye), 2) - ray_dot_ray * (dot(center_2_eye, center_2_eye) - pow(radius, 2));

    if (discriminant > 0) {
        discriminant = sqrt(discriminant);
        float init = -dot(ray, center_2_eye);
        float t1 = (init + discriminant) / ray_dot_ray;
        float t2 = (init - discriminant) / ray_dot_ray;

        float mint = min(t1, t2);
        if (mint < HIT_T_OFFSET) {
            return max(t1, t2);
        }
        return mint;
    }
    else if (discriminant == 0) {
        float init = -dot(ray, center_2_eye);
        float t1 = init / ray_dot_ray;
        return t1;
    }
    return MIN_T;
}

__global__
void textureCompute(hipSurfaceObject_t image)
{
    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    unsigned int x = threadIdx.x;
    unsigned int y = blockIdx.x;

    float3 sphere = make_float3(0.0, 0.0, -5.0);
    float3 eye = make_float3(0.0, 0.0, 0.0);
    float radius = 0.5;
    float3 ray = cast_ray(x, y, 512, 512) - eye;
    float sphereHit = check_hit_on_sphere(eye, ray, sphere, radius);
    uchar4 color;
    if (sphereHit >= 0 && sphereHit != MIN_T) {
        color = make_uchar4(255, 0, 0, 255);
    } else {
        color = make_uchar4(0, 0, 0, 255);
    }

    surf2Dwrite(color, image, x * sizeof(color), y, hipBoundaryModeClamp);
}

#define check(ans) { _check((ans), __FILE__, __LINE__); }
inline void _check(hipError_t code, char *file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

void MainCuda::texImageTest(Texture* texture) {
    struct hipGraphicsResource *vbo_res;
    // register this texture with CUDA
    //hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(),GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);
    check(hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    check(hipGraphicsMapResources(1, &vbo_res));

    hipArray_t viewCudaArray;
    check(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, vbo_res, 0, 0));

    hipResourceDesc viewCudaArrayResourceDesc;
    memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
    viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
    viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

    hipSurfaceObject_t viewCudaSurfaceObject;
    check(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));

    textureCompute<<<512, 512>>>(viewCudaSurfaceObject);

//    hipArray *array;
//    hipGraphicsMapResources(1, &vbo_res, 0);
//    hipGraphicsSubResourceGetMappedArray(&array, vbo_res, 0,0);
//
//    texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> texRef;
//    hipBindTextureToArray(texRef, (hipArray *)array));
//    texRef.filterMode = hipFilterModeLinear;
}

void MainCuda::doCalculation() {

    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Thread Blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;
    }

    glm::vec3 a;
    a.x = 3;
    std::cout << "A is: " << a.x << std::endl;

    int N = 1<<20; // 1M elements

    // Allocate Unified Memory -- accessible from CPU or GPU
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add<<<1, 1024>>>(N, x, y);
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    std::cout << y[0] << std::endl;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
}