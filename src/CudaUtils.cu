#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include "glm/glm.hpp"
#include "headers/CudaUtils.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <vector_functions.h>

//----------OPERATORS---------------------------------------------------------------------------------------------------

__device__ uchar4 operator+(const uchar4 &a, const uchar4 &b) {
    return make_uchar4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.z);
}

__device__ float3 operator*(const float3 &a, const float &b) {
    return make_float3(a.x*b, a.y*b, a.z*b);
}

__device__ float3 operator*(const float &a, const float3 &b) {
    return b * a;
}

__device__ float3 operator/(const float3 &a, const float &b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

__device__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ float3 operator*(const float3 &a, const float3 &b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

//----------VECTOR--OPERATIONS------------------------------------------------------------------------------------------

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ uchar4 toRGBA(const float3 &a) {
    return make_uchar4(int(a.x * 255), int(a.y * 255), int(a.z * 255), 255);
}

__device__ float3 t_to_vec(float3 e, float3 d, float t) {
    return e + (t * d);
}

__device__ float3 normalize(float3 a) {
    float mag = sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
    return make_float3(a.x, a.y, a.z)/mag;
}

//----------RT-FUNCTIONS------------------------------------------------------------------------------------------------

__device__ float3 cast_ray(unsigned int x, unsigned int y, int width, int height) {
    float d = 1.0;
    float fov = 60.0;
    float aspect_ratio = ((float)width) / ((float)height);
    float h = d * (float)tan((3.1415 * fov) / 180.0 / 2.0);
    float w = h * aspect_ratio;

    float top = h;
    float bottom = -h;
    float left = -w;
    float right = w;

    float u = left + (right - left) * float(x) / ((float)width);
    float v = bottom + (top - bottom) * (((float)height) - float(y)) / ((float)height);
    return make_float3(u, v, -d);
}

__device__ float3 getReflectedRay(float3 e, float3 d, float3 normal) {
    float3 ray_dir = normalize(d);
    return ray_dir - 2.0f * normal * dot(ray_dir, normal);
}

__device__ float3 getSphereNormal(float3 point, CudaSphere* sphere) {
    float3 normal = point - sphere->position;
    return normalize(normal);
}

__device__ float check_hit_on_sphere(float3 eye, float3 ray, float3 center, float radius) {
    float3 center_2_eye = eye - center;
    float ray_dot_ray = dot(ray, ray);
    float discriminant = pow(dot(ray, center_2_eye), 2) - ray_dot_ray * (dot(center_2_eye, center_2_eye) - pow(radius, 2));

    if (discriminant > 0) {
        discriminant = sqrt(discriminant);
        float init = -dot(ray, center_2_eye);
        float t1 = (init + discriminant) / ray_dot_ray;
        float t2 = (init - discriminant) / ray_dot_ray;

        float mint = min(t1, t2);
        if (mint < HIT_T_OFFSET) {
            return max(t1, t2);
        }
        return mint;
    }
    else if (discriminant == 0) {
        float init = -dot(ray, center_2_eye);
        float t1 = init / ray_dot_ray;
        return t1;
    }
    return MAX_T;
}

__device__ HitInfo doHitTest(float3 eye, float3 ray, CudaScene* scene) {
    HitInfo hit;
    for (int i=0; i<scene->numObjects; i++) {
        CudaSphere* sphere = (CudaSphere*)scene->objects[i];
        float sphereHit = check_hit_on_sphere(eye, ray, sphere->position, sphere->radius);
        if (sphereHit >= HIT_T_OFFSET && sphereHit < hit.t) {
            hit.object = sphere;
            hit.t = sphereHit;
            hit.hitPoint = t_to_vec(eye, ray, sphereHit);
            hit.index = i;
        }
    }
    return hit;
}

__device__ float3 traceSingleRay(float3 eye, float3 ray, CudaScene* scene, int bounceIndex, bool debug) {
    if (bounceIndex > 1) {
        //printf("Bounce greater than 1 ; %d", bounceIndex);
        return make_float3(0, 0, 0);
    }

    float3 color;
    HitInfo hitInfo = doHitTest(eye, ray, scene);
    if (hitInfo.isHit()) {
        float3 reflectedRay = normalize(getReflectedRay(eye, ray, getSphereNormal(hitInfo.hitPoint, (CudaSphere*)hitInfo.object)));

        if (debug) {
            printf("HitInfo(%d); Hit T(%f) @ (%f, %f, %f) - Reflected(%f, %f, %f)\n", hitInfo.index, hitInfo.t, hitInfo.hitPoint.x, hitInfo.hitPoint.y, hitInfo.hitPoint.z,
                   reflectedRay.x, reflectedRay.y, reflectedRay.z);
        }
        float3 reflectedRayColor = hitInfo.object->material->reflective * traceSingleRay(hitInfo.hitPoint, reflectedRay, scene, bounceIndex + 1, debug);
        color = hitInfo.object->material->diffuse + reflectedRayColor;
    } else {
        color = make_float3(0, 0, 0);
    }

    return color;
}

__global__ void kernel_traceRays(hipSurfaceObject_t image, CudaScene* scene)
{
    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    unsigned int x = threadIdx.x;
    unsigned int y = blockIdx.x;

    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, 512, 512) - eye;
    uchar4 color = toRGBA(traceSingleRay(eye, ray, scene, 0, false));

    surf2Dwrite(color, image, x * sizeof(color), y, hipBoundaryModeClamp);
}

__global__ void kernel_traceSingleRay(int x, int y, CudaScene* scene)
{
    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, 512, 512) - eye;
    printf("Ray (%f, %f, %f)\n", ray.x, ray.y, ray.z);
    traceSingleRay(eye, ray, scene, 0, true);
}

//----------------------------------------------------------------------------------------------------------------------
//---------------------------------------------Cuda Utils Class Definition----------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

#define check(ans) { _check((ans), __FILE__, __LINE__); }
inline void _check(hipError_t code, char *file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

CudaUtils::CudaUtils() {
}

CudaUtils::~CudaUtils() {

}

void CudaUtils::initializeRenderSurface(Texture* texture) {
    struct hipGraphicsResource *vbo_res;
    // register this texture with CUDA
    //hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(),GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);
    check(hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    check(hipGraphicsMapResources(1, &vbo_res));

    hipArray_t viewCudaArray;
    check(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, vbo_res, 0, 0));

    hipResourceDesc viewCudaArrayResourceDesc;
    memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
    viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
    viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

    check(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));
}

void CudaUtils::renderScene(CudaScene* cudaScene) {
    kernel_traceRays<<<512, 512>>>(CudaUtils::viewCudaSurfaceObject, cudaScene);
}

void CudaUtils::onClick(int x, int y, CudaScene* cudaScene) {
    kernel_traceSingleRay<<<1, 1>>>(x, y, cudaScene);
}

void CudaUtils::deviceInformation() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Thread Blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;

        hipDevice_t device;
        hipDeviceGet(&device, i);
        int major, minor;
        //hipDeviceComputeCapability(&major, &minor, device);
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
        std::cout << "Minor: " << minor << " \nMajor: " << major << std::endl;
    }

}

//----------------------------------------------------------------------------------------------------------------------
//---------------------------------------------Additional Utilities-----------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

float3 vec3ToFloat3(glm::vec3 vec) {
    return make_float3(vec.x, vec.y, vec.z);
}

template <class T>
T* cudaWrite(T* data, int len) {
    T* cudaPointer;
    check(hipMalloc((void**)&cudaPointer, sizeof(T) * len));
    check(hipMemcpy(cudaPointer, data, sizeof(T) * len, hipMemcpyHostToDevice));
    return cudaPointer;
}

template <class T>
T* cudaRead(T* src, int len) {
//    CudaRTObject** objs = new CudaRTObject*[1];
//    check(hipMemcpy(objs, cudaObjectsPtr, sizeof(CudaRTObject*), hipMemcpyDeviceToHost))

    T* hostPointer = (T*)malloc(len * sizeof(T));
    check(hipMemcpy(hostPointer, src, len * sizeof(T), hipMemcpyDeviceToHost))
    return hostPointer;
}

CudaMaterial* materialToCudaMaterial(Material* material) {
    CudaMaterial newMaterial(vec3ToFloat3(material->ambient), vec3ToFloat3(material->diffuse), vec3ToFloat3(material->specular),
                             material->shininess, vec3ToFloat3(material->reflective), vec3ToFloat3(material->transmissive),
                             material->refraction, material->roughness);
    return cudaWrite<CudaMaterial>(&newMaterial, 1);
}

CudaRTObject* rtObjectToCudaRTObject(RTObject* object) {
    switch (object->getType()) {
        case SPHERE:
            Sphere* sphere = (Sphere*)object;
            CudaSphere newSphere(vec3ToFloat3(sphere->getPosition()), sphere->getRadius(), materialToCudaMaterial(object->getMaterial()));
            return cudaWrite<CudaSphere>(&newSphere, 1);
    }
    return nullptr;
}

CudaScene* allocateCudaScene(Scene* scene) {
    int numObjects = scene->getObjects().size();
    auto objects = new CudaRTObject*[numObjects];
    int index = 0;
    for (RTObject* obj : scene->getObjects()) {
        CudaRTObject* cudaPtr = rtObjectToCudaRTObject(obj);
        if (cudaPtr != nullptr) {
            objects[index++] = cudaPtr;
        }
    }

    CudaRTObject** cudaObjectsPtr = cudaWrite<CudaRTObject *>(objects, index);
    CudaScene cudaScene(cudaObjectsPtr, index);
    return cudaWrite<CudaScene>(&cudaScene, 1);
}

BVHBinaryNode* createTreeHelper(std::vector<CudaTriangle*>* localTriangles, BVHBinaryNode* node) {
    int len = localTriangles->size();
    if (len <= 5) {
        int* indices = new int[len];
        for (int i=0; i<len; i++) {
            indices[i] = localTriangles->at(i)->index;
        }
        node->objectsIndex = indices;

        BVHBinaryNode tempNode(cudaWrite<Bounds>(node->bounds, 1), cudaWrite<int>(indices, len));
        return cudaWrite<BVHBinaryNode>(&tempNode, 1);
    }

    auto leftTriangles = new std::vector<CudaTriangle*>();
    auto rightTriangles = new std::vector<CudaTriangle*>();

    //bool xDiv, yDiv, zDiv;
    auto nb = *node->bounds;
    float xLen = nb.right - nb.left;
    float yLen = nb.top - nb.bottom;
    float zLen = nb.right - nb.left;
    if (xLen > yLen && xLen > zLen) {
        //xDiv = true;
        float mid = (nb.left + nb.right)/2;
        node->left = new BVHBinaryNode(new Bounds(nb.top, nb.bottom, nb.left, mid, nb.front, nb.back));
        node->right = new BVHBinaryNode(new Bounds(nb.top, nb.bottom, mid, nb.right, nb.front, nb.back));
    }
    else if (yLen > xLen && yLen > zLen) {
        //yDiv = true;
        float mid = (nb.top + nb.bottom)/2;
        node->left = new BVHBinaryNode(new Bounds(mid, nb.bottom, nb.left, nb.right, nb.front, nb.back));
        node->right = new BVHBinaryNode(new Bounds(nb.top, mid, nb.left, nb.right, nb.front, nb.back));
    }
    else if (zLen > yLen && zLen > xLen) {
        //zDiv = true;
        float mid = (nb.front + nb.back)/2;
        node->left = new BVHBinaryNode(new Bounds(nb.top, nb.bottom, nb.left, nb.right, mid, nb.back));
        node->right = new BVHBinaryNode(new Bounds(nb.top, nb.bottom, nb.left, nb.right, nb.front, mid));
    }

    for (CudaTriangle* t : *localTriangles) {
        //divide along the axis with max length
        if (isTriangleInBounds(t, node->left->bounds)) {
            leftTriangles->push_back(t);
        }
        else if (isTriangleInBounds(t, node->right->bounds)) {
            rightTriangles->push_back(t);
        }
    }

    BVHBinaryNode* leftNode = createTreeHelper(leftTriangles, node->left);
    delete leftTriangles;
    BVHBinaryNode* rightNode = createTreeHelper(rightTriangles, node->right);
    delete rightTriangles;

    BVHBinaryNode tempNode(cudaWrite<Bounds>(node->bounds, 1), leftNode, rightNode);
    return cudaWrite<BVHBinaryNode>(&tempNode, 1);
}

bool isTriangleInBounds(CudaTriangle* triangle, Bounds* bounds) {
    float3 pos = triangle->getPosition();
    return (pos.x > bounds->left && pos.x < bounds->right) &&
           (pos.y > bounds->bottom && pos.y < bounds->top) &&
           (pos.z > bounds->back && pos.z < bounds->front);
}

//    std::vector<CudaTriangle*> trianglesInBounds(std::vector<CudaTriangle*>* localTriangles, Bounds* bounds) {
//        auto leftTriangles = new std::vector<CudaTriangle*>();
//        for (CudaTriangle* t : *localTriangles) {
//
//        }
//    }

//    void createTreeHelper(CudaTriangle** localTriangles, int num, float3 position) {
//        for (int i=0; i<num; i++) {
//            CudaTriangle* t = localTriangles[i];
//            float3 position = t->getPosition();
//            if (position.y >= position.y) {
//                // If in the top half
//                if (position.x >= position.x) {
//                    // If in the top right half
//                    if (position.z >= position.z) {
//                        // If in the top right front
//                    } else {
//                        // If in the top right back
//                    }
//                } else {
//                    // If in the top left half
//                    if (position.z >= position.z) {
//                        // If in the top left front
//
//                    } else {
//                        // If in the top left back
//
//                    }
//                }
//            } else {
//                // If in the bottom half
//                if (position.x >= position.x) {
//                    // If in the bottom right half
//                    if (position.z >= position.z) {
//                        // If in the bottom right front
//                    } else {
//                        // If in the bottom right back
//                    }
//                } else {
//                    // If in the bottom left half
//                    if (position.z >= position.z) {
//                        // If in the bottom left front
//
//                    } else {
//                        // If in the bottom left back
//
//                    }
//                }
//            }
//        }
//    }

void cleanCudaScene(CudaScene* scene) {
    for (int i=0; i<scene->numObjects; i++) {
        hipFree(scene->objects[i]);
    }
    hipFree(scene);
}
