#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include "glm/glm.hpp"
#include "headers/CudaUtils.cuh"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <vector_functions.h>
#include <math_functions.h>

//----------OPERATORS---------------------------------------------------------------------------------------------------

__device__ __host__ uchar4 operator+(const uchar4 &a, const uchar4 &b) {
    return make_uchar4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float &b) {
    return make_float3(a.x*b, a.y*b, a.z*b);
}

__device__ __host__ float3 operator*(const float &a, const float3 &b) {
    return b * a;
}

__device__ __host__ float3 operator/(const float3 &a, const float &b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

__device__ __host__ float3 operator/(const float a, const float3 &b) {
    return make_float3(a/b.x, a/b.y, a/b.z);
}

__device__ __host__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ __host__ float3 operator-(const float3 &a, const float3 &b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

__device__ __host__ float3 operator*(const float3 &a, const float3 &b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

//----------VECTOR--OPERATIONS------------------------------------------------------------------------------------------

__device__ float dot(const float3 &a, const float3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ uchar4 toRGBA(const float3 &a) {
    return make_uchar4(int(a.x * 255), int(a.y * 255), int(a.z * 255), 255);
}

__device__ __host__ float3 t_to_vec(float3 e, float3 d, float t) {
    return e + (t * d);
}

__device__ float magnitude(float3 a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ float3 normalize(float3 a) {
    float mag = magnitude(a);
    return make_float3(a.x, a.y, a.z)/mag;
}

__device__ float3 cross(float3 a, float3 b) {
    return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

//----------RT-FUNCTIONS------------------------------------------------------------------------------------------------

__device__ float3 cast_ray(unsigned int x, unsigned int y, int width, int height) {
    float d = 1.0;
    float fov = 60.0;
    float aspect_ratio = ((float)width) / ((float)height);
    float h = d * (float)tan((3.1415 * fov) / 180.0 / 2.0);
    float w = h * aspect_ratio;

    float top = h;
    float bottom = -h;
    float left = -w;
    float right = w;

    float u = left + (right - left) * float(x) / ((float)width);
    float v = bottom + (top - bottom) * (((float)height) - float(y)) / ((float)height);
    return make_float3(u, v, -d);
}

__device__ float3 getReflectedRay(float3 e, float3 d, float3 normal) {
    float3 ray_dir = normalize(d);
    return ray_dir - 2.0f * normal * dot(ray_dir, normal);
}

__device__ float3 getSphereNormal(float3 point, CudaSphere* sphere) {
    float3 normal = point - sphere->position;
    return normalize(normal);
}

__device__ float checkHitOnPlane(float3 e, float3 d, float3 center, float3 normal) {
    /*Checks the hit on an infinite plane for the given normal and returns t value*/
    float denominator = dot(normal, d);
    if (denominator != 0.0) {
        float t = dot(normal, (center - e)) / denominator;
        return t;
    }
    return MAX_T;
}

__device__ float checkHitOnTriangle(float3 e, float3 d, float3 a, float3 b, float3 c) {
    /*Checks the hit on the triangle and returns t value. I first use the plane hit and then check if its inside the triangle*/
    float3 normal = normalize(cross(b - a, c - a));
    float t = checkHitOnPlane(e, d, a, normal);
    float3 x = t_to_vec(e, d, t);
    float aTest = dot(cross(b - a, x - a), normal);
    float bTest = dot(cross(c - b, x - b), normal);
    float cTest = dot(cross(a - c, x - c), normal);
    if (t != MAX_T && ((aTest >= 0 - HIT_T_OFFSET && bTest >= 0 - HIT_T_OFFSET && cTest >= 0 - HIT_T_OFFSET)
    || (aTest <= 0 + HIT_T_OFFSET && bTest <= 0 + HIT_T_OFFSET && cTest <= 0 + HIT_T_OFFSET))) {
        return t;
    }
    return MAX_T;
}

__device__ void printBounds(Bounds* bounds) {
    printf("AABB (%0.2f, %0.2f, %0.2f, %0.2f, %0.2f, %0.2f)", bounds->top, bounds->bottom,
           bounds->left, bounds->right, bounds->front, bounds->back);
}

__device__ void print2DUtil(BVHBinaryNode *root, int space)
{
    // Base case
    if (root == nullptr)
        return;

    // Increase distance between levels
    space += 10;

    // Process right child first
    print2DUtil(root->right, space);

    // Print current node after space
    // count
    printf("\n");
    for (int i = 10; i < space; i++)
        printf(" ");
    //print data
    printf("[{");
    //if (root->numObjects == 0) {
    //printBounds(root->bounds);
    //}
    printf("} ");
    for (int i=0; i<root->numObjects; i++) {
        printf("%d, ", root->objectsIndex[i]);
    }
    printf("]");
    //cout<<root->data<<"\n";

    // Process left child
    print2DUtil(root->left, space);
}

__device__ __host__ void swap(float &a, float &b) {
    float t = a;
    a = b;
    b = t;
}

/*
 * float3 invDir = 1.0 / d;
    //float3 invDir = make_float3(0.0, 0.0, 0.0) - d;

    float xVal = (invDir.x < 0) ? bounds->right : bounds->left;
    float yVal = (invDir.y < 0) ? bounds->top : bounds->bottom;
    float zVal = (invDir.z < 0) ? bounds->front : bounds->back;
    float tmin, tmax, tymin, tymax, tzmin, tzmax;

    tmin = (xVal - e.x) * invDir.x;
    tmax = (xVal - e.x) * invDir.x;
    tymin = (yVal - e.y) * invDir.y;
    tymax = (yVal - e.y) * invDir.y;

    if ((tmin > tymax) || (tymin > tmax))
        return MAX_T;
    if (tymin > tmin)
        tmin = tymin;
    if (tymax < tmax)
        tmax = tymax;

    tzmin = (zVal - e.z) * invDir.z;
    tzmax = (zVal - e.z) * invDir.z;

    if ((tmin > tzmax) || (tzmin > tmax))
        return MAX_T;
    if (tzmin > tmin)
        tmin = tzmin;
    if (tzmax < tmax)
        tmax = tzmax;
 */

__device__ __host__ float checkHitOnAABB(float3 e, float3 d, Bounds* bounds, bool debug) {
    float tmin = (bounds->left - e.x) / d.x;
    float tmax = (bounds->right - e.x) / d.x;

    if (tmin > tmax) swap(tmin, tmax);

    float tymin = (bounds->bottom - e.y) / d.y;
    float tymax = (bounds->top - e.y) / d.y;

    if (tymin > tymax) swap(tymin, tymax);

    if ((tmin > tymax) || (tymin > tmax))
        return MAX_T;

    if (tymin > tmin)
        tmin = tymin;

    if (tymax < tmax)
        tmax = tymax;

    float tzmin = (bounds->back - e.z) / d.z;
    float tzmax = (bounds->front - e.z) / d.z;

    if (tzmin > tzmax) swap(tzmin, tzmax);

    if ((tmin > tzmax) || (tzmin > tmax))
        return MAX_T;

    if (tzmin > tmin)
        tmin = tzmin;

    if (tzmax < tmax)
        tmax = tzmax;

    return tmin;
}

__device__ float checkHitOnMeshHelper(float3 eye, float3 ray, BVHBinaryNode* node, CudaMesh* mesh, bool debug) {
    float minT = MAX_T;
    if (node->numObjects != 0) { // Is a leaf node
        for (int j=0; j<node->numObjects; j++) {
            CudaTriangle t = mesh->triangles[node->objectsIndex[j]];
            float triangleHit = checkHitOnTriangle(eye, ray, t.a, t.b, t.c);
            if(debug) {
                printf("Checking hits on triangle (%d) -- (%f)\n", node->objectsIndex[j], triangleHit);
            }
            if (triangleHit < minT) {
                minT = triangleHit;
                if (debug) {
                    printf("New hit on triangle at (%d) MinT (%f)\n", node->objectsIndex[j], minT);
                }
            }
        }
    }

    if (node->left == nullptr || node->right == nullptr) {
        return minT;
    }

    float leftT = checkHitOnAABB(eye, ray, node->left->bounds, debug);
    float rightT = checkHitOnAABB(eye, ray, node->right->bounds, debug);

//    if (debug) {
//        printf("LeftT - AABB (%f, %f, %f, %f, %f, %f) ---- (%f)\n", node->left->bounds->top, node->left->bounds->bottom,
//               node->left->bounds->left, node->left->bounds->right, node->left->bounds->front, node->left->bounds->back, leftT);
//
//        printf("RightT - AABB (%f, %f, %f, %f, %f, %f) ---- (%f)\n", node->right->bounds->top, node->right->bounds->bottom,
//               node->right->bounds->left, node->right->bounds->right, node->right->bounds->front, node->right->bounds->back, rightT);
//    }

    if (leftT != MAX_T && leftT <= minT) {
        if (debug) {
            printf("Checking left LeftT(%f) MinT(%f)\n", leftT, minT);
        }
        float tempT = checkHitOnMeshHelper(eye, ray, node->left, mesh, debug);
        minT = min(tempT, minT);
    }
    if (rightT != MAX_T && rightT <= minT) {
        if (debug) {
            printf("Checking right RightT(%f) MinT(%f)\n", rightT, minT);
        }
        float tempT = checkHitOnMeshHelper(eye, ray, node->right, mesh, debug);
        minT = min(tempT, minT);
    }

    if (debug) {
        printf("Returning MinT(%f)\n", minT);
    }
    return minT;
}

__device__ float checkHitOnMesh(float3 eye, float3 ray, BVHBinaryNode* node, CudaMesh* mesh, bool debug) {
    float t = checkHitOnAABB(eye, ray, node->bounds, debug);
    if (debug) {
        printf("Main AABB Hit @ (%f)\n", t);
        printf("\n\n");
        print2DUtil(node, 0);
        printf("\n\n");

    }

    if (t != MAX_T) { // If node is hit
        return checkHitOnMeshHelper(eye, ray, node, mesh, debug);
    } else {
        return MAX_T;
    }
}

__device__ float check_hit_on_sphere(float3 eye, float3 ray, float3 center, float radius) {
    float3 center_2_eye = eye - center;
    float ray_dot_ray = dot(ray, ray);
    float discriminant = pow(dot(ray, center_2_eye), 2) - ray_dot_ray * (dot(center_2_eye, center_2_eye) - pow(radius, 2));

    if (discriminant > 0) {
        discriminant = sqrt(discriminant);
        float init = -dot(ray, center_2_eye);
        float t1 = (init + discriminant) / ray_dot_ray;
        float t2 = (init - discriminant) / ray_dot_ray;

        float mint = min(t1, t2);
        if (mint < HIT_T_OFFSET) {
            return max(t1, t2);
        }
        return mint;
    }
    else if (discriminant == 0) {
        float init = -dot(ray, center_2_eye);
        float t1 = init / ray_dot_ray;
        return t1;
    }
    return MAX_T;
}

__device__ HitInfo doHitTest(float3 eye, float3 ray, CudaScene* scene, bool debug) {
    HitInfo hit;
    for (int i=0; i<scene->numObjects; i++) {
        if (scene->objects[i]->type == SPHERE) {
            CudaSphere* sphere = (CudaSphere*)scene->objects[i];
            float sphereHit = check_hit_on_sphere(eye, ray, sphere->position, sphere->radius);
            if (sphereHit >= HIT_T_OFFSET && sphereHit < hit.t) {
                hit.object = sphere;
                hit.t = sphereHit;
                hit.hitPoint = t_to_vec(eye, ray, sphereHit);
                hit.index = i;

                if (debug) {
                    printf("doHitTest @ index (%d) with t (%f)\n", i, sphereHit);
                }
            }
        }
        else if (scene->objects[i]->type == MESH) {
            CudaMesh* mesh = (CudaMesh*)scene->objects[i];
            float meshHit = checkHitOnMesh(eye, ray, mesh->bvhRoot, mesh, debug);
            if (meshHit >= HIT_T_OFFSET && meshHit < hit.t) {
                hit.object = mesh;
                hit.t = meshHit;
                hit.hitPoint = t_to_vec(eye, ray, meshHit);
                hit.index = i;
                if (debug) {
                    printf("doHitTest @ index (%d) with t (%f)\n", i, meshHit);
                }
            }
//            if (debug) {
//                for (int k=0; k<mesh->numTriangles; k++) {
//                    CudaTriangle tt = mesh->triangles[k];
//                    printf("Index(%d); (%f, %f, %f)\n", k, tt.a.x, tt.b.x, tt.c.x);
//                }
//            }
//            for (int j=0; j<mesh->numTriangles; j++) {
//                CudaTriangle t = mesh->triangles[j];
//                float triangleHit = checkHitOnTriangle(eye, ray, t.a, t.b, t.c);
//                if (triangleHit >= HIT_T_OFFSET && triangleHit < hit.t) {
//                    hit.object = mesh;
//                    hit.t = triangleHit;
//                    hit.hitPoint = t_to_vec(eye, ray, triangleHit);
//                    hit.index = i;
//                }
//            }
        }
    }
    return hit;
}

__device__ float3 traceSingleRay(float3 eye, float3 ray, CudaScene* scene, int bounceIndex, int maxBounces, bool debug) {
    if (bounceIndex >= maxBounces) {
        //printf("Bounce greater than 1 ; %d", bounceIndex);
        return make_float3(0, 0, 0);
    }

    float3 color;
    HitInfo hitInfo = doHitTest(eye, ray, scene, debug);
    if (hitInfo.isHit()) {
        float3 reflectedRay = normalize(getReflectedRay(eye, ray, getSphereNormal(hitInfo.hitPoint, (CudaSphere*)hitInfo.object)));
        if (debug) {
            printf("HitInfo(%d); Hit T(%f) @ (%f, %f, %f) - Reflected(%f, %f, %f)\n", hitInfo.index, hitInfo.t, hitInfo.hitPoint.x, hitInfo.hitPoint.y, hitInfo.hitPoint.z,
                   reflectedRay.x, reflectedRay.y, reflectedRay.z);
        }
        float3 reflectedRayColor = hitInfo.object->material->reflective * traceSingleRay(hitInfo.hitPoint, reflectedRay, scene,bounceIndex + 1, maxBounces, debug);
        color = hitInfo.object->material->diffuse + reflectedRayColor;

    } else {
        color = make_float3(0, 0, 0);
    }

    return color;
}

__global__ void kernel_traceRays(hipSurfaceObject_t image, CudaScene* scene)
{
    // blockIdx - index of block in grid
    // theadIdx - index of thread in block
    unsigned int x = threadIdx.x;
    unsigned int y = blockIdx.x;

    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, 512, 512) - eye;
    uchar4 color = toRGBA(traceSingleRay(eye, ray, scene, 0, 2, false));

    surf2Dwrite(color, image, x * sizeof(color), 512-y, hipBoundaryModeClamp);
}

__global__ void kernel_traceSingleRay(hipSurfaceObject_t image, int x, int y, CudaScene* scene)
{
    float3 eye = make_float3(0.0, 0.0, 0.0);
    float3 ray = cast_ray(x, y, 512, 512) - eye;
    printf("Ray (%f, %f, %f)\n", ray.x, ray.y, ray.z);
    uchar4 color = toRGBA(traceSingleRay(eye, ray, scene, 0, 4, true));
    printf("Final Color: (%d, %d, %d, %d)\n", color.x, color.y, color.z, color.w);
    surf2Dwrite(color, image, x * sizeof(color), 512-y, hipBoundaryModeClamp);
}

//----------------------------------------------------------------------------------------------------------------------
//---------------------------------------------Cuda Utils Class Definition----------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

CudaUtils::CudaUtils() {
}

CudaUtils::~CudaUtils() {

}

void CudaUtils::initializeRenderSurface(Texture* texture) {
    struct hipGraphicsResource *vbo_res;
    // register this texture with CUDA
    //hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(),GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly);
    check(hipGraphicsGLRegisterImage(&vbo_res, texture->getTextureId(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
    check(hipGraphicsMapResources(1, &vbo_res));

    hipArray_t viewCudaArray;
    check(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, vbo_res, 0, 0));

    hipResourceDesc viewCudaArrayResourceDesc;
    memset(&viewCudaArrayResourceDesc, 0, sizeof(viewCudaArrayResourceDesc));
    viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
    viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

    check(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));
}

void CudaUtils::renderScene(CudaScene* cudaScene) {
    kernel_traceRays<<<512, 512>>>(CudaUtils::viewCudaSurfaceObject, cudaScene);
    check(hipDeviceSynchronize());
    //test hits
//    Bounds* test = new Bounds(0.5, -0.5, -0.5, 0.5, -2.0, -3.0);
//    std::cout << "AABB HIT: " << checkHitOnAABB(make_float3(0.0, 0.0, 0.0), make_float3(0.0, 0.0, -1.0), test) << std::endl;
}

void CudaUtils::onClick(int x, int y, CudaScene* cudaScene) {
    kernel_traceSingleRay<<<1, 1>>>(CudaUtils::viewCudaSurfaceObject, x, y, cudaScene);
    check(hipDeviceSynchronize());
}

void CudaUtils::deviceInformation() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Max Thread Blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
        std::cout << "  Max Threads per block: " << prop.maxThreadsPerBlock << std::endl;

        hipDevice_t device;
        hipDeviceGet(&device, i);
        int major, minor;
        //hipDeviceComputeCapability(&major, &minor, device);
        hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
        hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
        std::cout << "Minor: " << minor << " \nMajor: " << major << std::endl;
    }
}

